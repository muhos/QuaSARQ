#include "hip/hip_runtime.h"

#include "simulator.hpp"
#include "datatypes.cuh"
#include "shared.cuh"
#include "print.cuh"
#include "tuner.cuh"
#include "grid.cuh"


namespace QuaSARQ {


    INLINE_DEVICE size_t compute_block_index(const size_t& i, const size_t& j, const size_t& w, const size_t& num_words_major) {
        return ((i << WORD_POWER) + j) * num_words_major + w;
    }

    NOINLINE_DEVICE void transpose_tile(word_std_t* data, word_std_t* tile, const size_t& tile_offset) {
        assert(blockDim.x == WORD_BITS);

        static const word_std_t masks[WORD_POWER] = {
#if defined(WORD_SIZE_8)
            0x55U,  // separate odds/evens
            0x33U,  // separate bit pairs
            0x0FU
#elif defined(WORD_SIZE_32)
            0x55555555ULL,  // separate odds/evens
            0x33333333ULL,  // separate bit pairs
            0x0F0F0F0FULL,
            0x00FF00FFULL,
            0x0000FFFFULL
#elif defined(WORD_SIZE_64)
            0x5555555555555555ULL,  // separate odds/evens
            0x3333333333333333ULL,  // separate bit pairs
            0x0F0F0F0F0F0F0F0FULL,
            0x00FF00FF00FF00FFULL,
            0x0000FFFF0000FFFFULL,
            0x00000000FFFFFFFFULL
#endif
        };

        static const uint32 offsets[WORD_POWER] = { 
#if defined(WORD_SIZE_8)
            1, 2, 4
#elif defined(WORD_SIZE_32)
            1, 2, 4, 8, 16
#elif defined(WORD_SIZE_64)
            1, 2, 4, 8, 16, 32
#endif
        };

        uint32 tid = threadIdx.x;
        uint32 shared_tid = threadIdx.y * blockDim.x + tid;
        tile[shared_tid] = data[tid * tile_offset];
        __syncthreads();

        #pragma unroll
        for (int pairs = 0; pairs < WORD_POWER; pairs++) {
            const word_std_t mask = masks[pairs];
            const word_std_t imask = ~mask;
            const uint32 offset = offsets[pairs];
            if (!(tid & offset)) {
                word_std_t& x = tile[shared_tid];
                word_std_t& y = tile[shared_tid + offset];
                word_std_t a = x & mask;
                word_std_t b = x & imask;
                word_std_t c = y & mask;
                word_std_t d = y & imask;
                x = a | (c << offset);
                y = (b >> offset) | d;
            }
            __syncthreads(); // ensure all threads see the updated tile before next pairs
        }

        data[tid * tile_offset] = tile[shared_tid];
    }

    __global__ void transpose_tiles_kernel(Table* xs, Table* zs, const size_t num_words_major, const size_t num_words_minor, const bool row_major) {
        assert(gridDim.z == 2);
        word_std_t* data = reinterpret_cast<word_std_t*>((!blockIdx.z) ? xs->data() : zs->data());
        word_std_t* shared = SharedMemory<word_std_t>();
        for_parallel_y(a, num_words_minor) {
            for (size_t b = blockIdx.x; b < num_words_major; b += gridDim.x) {
                // Inline transpose a tile of WORD_BITS words, each word has WORD_BITS bits.
                // Transposition is done in shared memory.
                const size_t tile_index = compute_block_index(a, 0, b, num_words_major);
                transpose_tile(data + tile_index, shared, num_words_major);
            }
        }
    }

    INLINE_DEVICE void swap_tile(word_std_t* data, word_std_t* shared, const size_t& a, const size_t& b, const size_t& num_words_major, const size_t& offset) {
        word_std_t* above_diagonal = shared;
        word_std_t* below_diagonal = shared + blockDim.y * blockDim.x;
        int tid = threadIdx.x;
        int shared_tid = threadIdx.y * blockDim.x + threadIdx.x;
        const size_t a_idx = compute_block_index(a, tid, b + offset, num_words_major);
        const size_t b_idx = compute_block_index(b, tid, a + offset, num_words_major);
        above_diagonal[shared_tid] = data[a_idx];
        below_diagonal[shared_tid] = data[b_idx];
        __syncthreads();
        data[a_idx] = below_diagonal[shared_tid];
        data[b_idx] = above_diagonal[shared_tid];
    }

    __global__ void swap_tiles_kernel(Table* xs, Table* zs, const size_t num_words_major, const size_t num_words_minor) {
        assert(gridDim.z == 2);
        word_std_t* data = reinterpret_cast<word_std_t*>((!blockIdx.z) ? xs->data() : zs->data());
        word_std_t* shared = SharedMemory<word_std_t>();
        for_parallel_y(a, num_words_minor) {
            for (size_t b = blockIdx.x; b < num_words_minor; b += gridDim.x) {
                // Only swap words above diagonal
                if (b > a) {
                    // Do the destabilizers.
                    swap_tile(data, shared, a, b, num_words_major, 0);
                    // Do the stabilizers.
                    swap_tile(data, shared, a, b, num_words_major, num_words_minor);
                }
            }
        }
    }

    __global__ void rowmajor_kernel(Table *xs, Table *zs,
                                    const size_t num_words_major,
                                    const size_t num_words_minor,
                                    const size_t num_qubits_padded,
                                    const bool row_major)
    {
        assert(gridDim.z == 2);
        Table& t = (!blockIdx.z) ? *xs : *zs;
        if (!blockIdx.x && !blockIdx.y && !threadIdx.x) {
            t.flag_orientation(row_major);
        }
        word_std_t* shared = SharedMemory<word_std_t>();
        word_std_t* tile_destab = shared;
        word_std_t* tile_stab = tile_destab + blockDim.y * blockDim.x;
        size_t tile_idx = threadIdx.y * blockDim.x + threadIdx.x;
        grid_t q = blockIdx.x * blockDim.x + threadIdx.x; 
        grid_t w = blockIdx.y * blockDim.y + threadIdx.y; 
        if (q < num_qubits_padded && w < num_words_minor) {
            const size_t old_idx = row_major ? q * num_words_major + w : w * num_qubits_padded + q;
            const size_t old_stab_offset = row_major ? num_words_minor : num_qubits_padded * num_words_minor;
            tile_destab[tile_idx] = t[old_idx];
            tile_stab  [tile_idx] = t[old_idx + old_stab_offset];
            __syncthreads();
            const size_t new_idx = row_major ? w * num_qubits_padded + q : q * num_words_major + w;
            const size_t new_stab_offset = row_major ? num_qubits_padded * num_words_minor : num_words_minor;
            t[new_idx] = tile_destab[tile_idx];
            t[new_idx + new_stab_offset] = tile_stab[tile_idx];
        }
    }

    void Simulator::transpose(const bool& row_major, const hipStream_t& stream) {
        const size_t num_words_minor = tableau.num_words_minor();
        const size_t num_words_major = tableau.num_words_major();
        dim3 currentblock, currentgrid;

        if (options.tune_transposebits || options.tune_transposeswap) {
            SYNCALL;
            tune_inplace_transpose(transpose_tiles_kernel, swap_tiles_kernel, 
            bestblocktransposebits, bestgridtransposebits, 
            bestblocktransposeswap, bestgridtransposeswap, 
            XZ_TABLE(tableau), num_words_major, num_words_minor, row_major);
        }
        if (options.tune_transposerowmajor) {
            SYNCALL;
            tune_rowmajor_transpose(rowmajor_kernel, 
            bestblocktransposerowmajor, bestgridtransposerowmajor, 
            XZ_TABLE(tableau), num_words_major, num_words_minor, tableau.num_qubits_padded(), row_major);
        }

        bestgridtransposerowmajor.z = 2;
        if (bestgridtransposerowmajor.x == 1)
            bestgridtransposerowmajor.x = ROUNDUP(tableau.num_qubits_padded(), bestblocktransposerowmajor.x);
        if (bestgridtransposerowmajor.y == 1)
            bestgridtransposerowmajor.y = ROUNDUP(num_words_minor, bestblocktransposerowmajor.y);
        OPTIMIZESHARED(rowmajor_smem_size, bestblocktransposerowmajor.y * bestblocktransposerowmajor.x, 2 * sizeof(word_std_t));
        
        if (!row_major) {
            currentblock = bestblocktransposerowmajor, currentgrid = bestgridtransposerowmajor;
            LOGN2(2, "Running column-major with block(x:%u, y:%u) and grid(x:%u, y:%u, z:%u).. ", currentblock.x, currentblock.y, currentgrid.x, currentgrid.y, currentgrid.z);
            rowmajor_kernel <<<currentgrid, currentblock, rowmajor_smem_size, stream>>> (
                XZ_TABLE(tableau), 
                num_words_major, 
                num_words_minor, 
                tableau.num_qubits_padded(),
                row_major
            );
            if (options.sync) {
                LASTERR("failed to launch rowmajor kernel");
                SYNC(stream);
            }
            LOGDONE(2, 4);
        }

        bestblocktransposebits.x = WORD_BITS;
        bestgridtransposebits.x = MIN(num_words_major, bestgridtransposebits.x); 
        bestgridtransposebits.z = 2;
        TRIM_Y_BLOCK_IN_DEBUG_MODE(bestblocktransposebits, bestgridtransposebits, num_words_minor);
        TRIM_GRID_IN_2D(bestblocktransposebits, bestgridtransposebits, num_words_minor, y);
        currentblock = bestblocktransposebits, currentgrid = bestgridtransposebits;
        OPTIMIZESHARED(transpose_smem_size, currentblock.y * currentblock.x, sizeof(word_std_t));
        LOGN2(2, "Running transpose-tiles with block(x:%u, y:%u) and grid(x:%u, y:%u, z:%u).. ", currentblock.x, currentblock.y, currentgrid.x, currentgrid.y, currentgrid.z);
        transpose_tiles_kernel << <currentgrid, currentblock, transpose_smem_size, stream >> > (XZ_TABLE(tableau), num_words_major, num_words_minor, row_major);
        if (options.sync) {
            LASTERR("failed to launch transpose-tiles kernel");
            SYNC(stream);
        }
        LOGDONE(2, 4);

        bestblocktransposeswap.x = WORD_BITS;
        bestgridtransposeswap.x = MIN(num_words_minor, bestgridtransposeswap.x); 
        bestgridtransposeswap.z = 2;
        TRIM_Y_BLOCK_IN_DEBUG_MODE(bestblocktransposeswap, bestgridtransposeswap, num_words_minor);
        TRIM_GRID_IN_2D(bestblocktransposeswap, bestgridtransposeswap, num_words_minor, y);
        currentblock = bestblocktransposeswap, currentgrid = bestgridtransposeswap;
        OPTIMIZESHARED(swap_smem_size, currentblock.y * currentblock.x, 2 * sizeof(word_std_t));
        LOGN2(2, "Running swap-tiles with block(x:%u, y:%u) and grid(x:%u, y:%u, z:%u).. ", currentblock.x, currentblock.y, currentgrid.x, currentgrid.y, currentgrid.z);
        swap_tiles_kernel << <currentgrid, currentblock, swap_smem_size, stream >> > (XZ_TABLE(tableau), num_words_major, num_words_minor);
        if (options.sync) {
            LASTERR("failed to launch swap-tiles kernel");
            SYNC(stream);
        }
        LOGDONE(2, 4);

        if (row_major) {
            //print_tableau(tableau, -1, false); 
            currentblock = bestblocktransposerowmajor, currentgrid = bestgridtransposerowmajor;
            LOGN2(2, "Running row-major with block(x:%u, y:%u) and grid(x:%u, y:%u, z:%u).. ", currentblock.x, currentblock.y, currentgrid.x, currentgrid.y, currentgrid.z);
            rowmajor_kernel <<<currentgrid, currentblock, rowmajor_smem_size, stream>>> (
                XZ_TABLE(tableau), 
                num_words_major, 
                num_words_minor, 
                tableau.num_qubits_padded(),
                row_major
            );
            if (options.sync) {
                LASTERR("failed to launch rowmajor kernel");
                SYNC(stream);
            }
            LOGDONE(2, 4);
        }
        //print_tableau(tableau, -1, false);        
        
    }

}

