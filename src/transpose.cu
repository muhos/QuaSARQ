#include "hip/hip_runtime.h"

#include "simulator.hpp"
#include "datatypes.cuh"
#include "print.cuh"
#include "tuner.cuh"
#include "grid.cuh"


namespace QuaSARQ {


    __device__ inline size_t compute_block_index(const size_t& i, const size_t& j, const size_t& w, const size_t& num_words_major) {
        return ((i << WORD_POWER) + j) * num_words_major + w;
    }

    __device__ void shared_inplace_transpose(word_std_t* data, size_t stride)
    {

        assert(blockDim.x == WORD_BITS);

        static const word_std_t  masks[WORD_POWER] = {
#if defined(WORD_SIZE_8)
            0x55U,  // separate odds/evens
            0x33U,  // separate bit pairs
            0x0FU
#elif defined(WORD_SIZE_32)
            0x55555555ULL,  // separate odds/evens
            0x33333333ULL,  // separate bit pairs
            0x0F0F0F0FULL,
            0x00FF00FFULL,
            0x0000FFFFULL
#elif defined(WORD_SIZE_64)
            0x5555555555555555ULL,  // separate odds/evens
            0x3333333333333333ULL,  // separate bit pairs
            0x0F0F0F0F0F0F0F0FULL,
            0x00FF00FF00FF00FFULL,
            0x0000FFFF0000FFFFULL,
            0x00000000FFFFFFFFULL
#endif
        };

        static const unsigned shifts[WORD_POWER] = { 
#if defined(WORD_SIZE_8)
            1, 2, 4
#elif defined(WORD_SIZE_32)
            1, 2, 4, 8, 16
#elif defined(WORD_SIZE_64)
            1, 2, 4, 8, 16, 32
#endif
        };

        // We'll store all WORD_BITS lines in shared memory
        // tile[k] will hold the row k (one WORD_BITS-bit word)
        __shared__ word_std_t tile[WORD_BITS];

        // 1) Load from global to shared
        size_t tid = threadIdx.x;

        tile[tid] = data[tid * stride];

        __syncthreads();

#pragma unroll
        for (int pass = 0; pass < WORD_POWER; pass++) {
            word_std_t mask = masks[pass];
            word_std_t imask = ~mask;
            unsigned shift = shifts[pass];

            if ((tid & shift) == 0) {
                word_std_t& x = tile[tid];
                word_std_t& y = tile[tid + shift];

                word_std_t a = x & mask;
                word_std_t b = x & imask;
                word_std_t c = y & mask;
                word_std_t d = y & imask;

                x = a | (c << shift);
                y = (b >> shift) | d;
            }

            __syncthreads(); // ensure all threads see the updated tile before next pass
        }

        // 3) Write back from shared to global
        data[tid * stride] = tile[tid];
    }

    template <word_std_t mask, word_std_t shift>
    INLINE_DEVICE void inplace_transpose_8x8_pass(word_std_t* data, size_t stride) {
        for (size_t k = 0; k < WORD_BITS; k++) {
            if (k & shift) {
                continue;
            }
            word_std_t& x = data[stride * k];
            word_std_t& y = data[stride * (k + shift)];
            word_std_t a = x & mask;
            word_std_t b = x & ~mask;
            word_std_t c = y & mask;
            word_std_t d = y & ~mask;
            x = a | (c << shift);
            y = (b >> shift) | d;
        }
    }

    INLINE_DEVICE void inplace_transpose_8x8(word_std_t* data, size_t stride) {
        inplace_transpose_8x8_pass<0x55UL, 1>(data, stride);
        inplace_transpose_8x8_pass<0x33UL, 2>(data, stride);
        inplace_transpose_8x8_pass<0x0FUL, 4>(data, stride);
    }

    __global__ void transpose_kernel(Table* xs, Table* zs, size_t num_words_major, size_t num_words_minor)
    {
        word_std_t* data = reinterpret_cast<word_std_t*>(zs->data());
        if (!blockIdx.x && !blockIdx.y && !threadIdx.x)
            zs->flag_rowmajor();
        for (size_t a = blockIdx.y; a < num_words_minor; a += gridDim.y) {
            for (size_t b = blockIdx.x; b < num_words_minor; b += gridDim.x) {
                // Inline transpose a tile of WORD_BITS words, each word has WORD_BITS bits.
                // Transposition is done in shared memory.
                // tile_index = (a << WORD_POWER) * num_words_major + b + num_words_minor;
                const size_t tile_index = compute_block_index(a, 0, b + num_words_minor, num_words_major);
                shared_inplace_transpose(data + tile_index, num_words_major);
                //inplace_transpose_8x8(data + tile_index, num_words_major);
                //print_table(*xs, true);
            }
        }
    }

    /*
0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  1  0  0  0  0  0  0    0  0  0  0  0  0  0  0    
1    1  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    1  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    
2    0  0  1  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  1  0  0  0  0  0    0  0  0  0  0  0  0  0    
3    1  0  0  1  0  0  0  0    0  0  0  0  0  0  0  0    1  0  0  0  0  1  0  0    0  0  0  0  0  0  0  0    
4    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  1  0  0  0    0  0  0  0  0  0  0  0    
5    0  0  0  1  0  1  0  0    0  0  0  0  0  0  0  0    0  0  0  1  0  0  0  0    0  0  0  0  0  0  0  0    
6    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    1  0  0  0  0  0  0  0    
7    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  1  0  0  1    0  0  0  0  0  0  0  0    
8    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  1  0    0  0  0  0  0  0  0  0    
9    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  1  0  0  0  0  0  0    
10   0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    
11   0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    
12   0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    
13   0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    
14   0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    
15   0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0    0  0  0  0  0  0  0  0   
    */

    __global__ void swap_kernel(word_std_t* data, size_t num_words_major, size_t num_words_minor)
    {
        //assert(blockDim.x == WORD_BITS);

        // Shared memory to hold rows for sub-block (a,b) and (b,a).
        __shared__ word_std_t tileA[WORD_BITS];
        __shared__ word_std_t tileB[WORD_BITS];

        for (size_t a = blockIdx.y; a < num_words_minor; a += gridDim.y) {
            // Only swap if b > a (above-diagonal)
            for (size_t b = blockIdx.x; b < num_words_minor; b += gridDim.x) {
                if (b > a) {
                    // for (size_t k = 0; k < WORD_BITS; k++) {
                    //     // ((a << WORD_POWER) + k) * num_words_major + b;
                    //     size_t a_idx = compute_block_index(a, k, b + num_words_minor, num_words_major);
                    //     size_t b_idx = compute_block_index(b, k, a + num_words_minor, num_words_major);
                    //     word_std_t tmp = data[a_idx];
                    //     data[a_idx] = data[b_idx];
                    //     data[b_idx] = tmp;

                    // }

                    // We have WORD_BITS threads in x, each tid handles one 'maj_low' in [0..63].
                    int tid = threadIdx.x;

                    // 1) Load WORD_BITS words from global memory into tileA and tileB
                    // ((a << WORD_POWER) + tid) * num_words_major + b;
                    size_t a_idx = compute_block_index(a, tid, b + num_words_minor, num_words_major);
                    size_t b_idx = compute_block_index(b, tid, a + num_words_minor, num_words_major);

                    tileA[tid] = data[a_idx];
                    tileB[tid] = data[b_idx];

                    __syncthreads();

                    // 2) Swap: the data from (a, tid, b) should go to (b, tid, a), and vice versa.
                    //    Actually we've already loaded them, so we just need to write them back swapped.

                    // 3) Write swapped data back to global memory
                    data[a_idx] = tileB[tid];
                    data[b_idx] = tileA[tid];
                }
            }
        }
    }

    __global__ void transpose_to_rowmajor(Table *inv_xs, Table *inv_zs, Signs *inv_ss,
                              const Table *  xs, const Table *  zs, const Signs *  ss,
                              const size_t num_words_major, const size_t num_words_minor,
                              const size_t num_qubits) {
        if (!global_ty) {
            for_parallel_x(w, num_words_major) {
                sign_t signs_word = (*ss)[w];
                const size_t word_idx = w * WORD_BITS;
                #pragma unroll
                for (uint32 j = 0; j < WORD_BITS; j++) {
                    inv_ss->unpacked_data()[word_idx + j] = ((signs_word >> j) & 1) * 2;
                }
            }
            if (!global_tx) {
                inv_xs->flag_rowmajor();
                inv_zs->flag_rowmajor();
            }
        }

        for_parallel_y(w, 2 * num_qubits) {
            const word_std_t generator_index_per_word = (w & WORD_MASK);
            for_parallel_x(q, num_words_minor) {
                word_std_t inv_word_x = 0;
                word_std_t inv_word_z = 0;
                const size_t block_idx = q * WORD_BITS * num_words_major + WORD_OFFSET(w);
                #pragma unroll
                for (uint32 k = 0; k < WORD_BITS; k++) {
                    const size_t src_word_idx = k * num_words_major + block_idx;
                    const word_std_t generators_word_x = (*xs)[src_word_idx];
                    const word_std_t generators_word_z = (*zs)[src_word_idx];
                    const word_std_t generator_bit_x = (generators_word_x >> generator_index_per_word) & 1;
                    const word_std_t generator_bit_z = (generators_word_z >> generator_index_per_word) & 1;
                    inv_word_x |= (generator_bit_x << k);
                    inv_word_z |= (generator_bit_z << k);
                }
                const size_t dest_word_idx = q + w * num_words_minor;
                (*inv_xs)[dest_word_idx] = inv_word_x;
                (*inv_zs)[dest_word_idx] = inv_word_z;
            }
        }
    }

    __global__ void transpose_to_colmajor(Table* xs, Table* zs, Signs* ss, 
                        ConstTablePointer inv_xs, ConstTablePointer inv_zs, ConstSignsPointer  inv_ss, 
                        const size_t num_words_major, const size_t num_words_minor, 
                        const size_t num_qubits) {

        if (!global_ty) {
            sign_t *packed_signs = ss->data();
            for_parallel_x(w, num_words_major) {
                sign_t signs_word = 0;
                const size_t word_idx = w * WORD_BITS;
                #pragma unroll
                for (uint32 j = 0; j < WORD_BITS; j++) {
                    sign_t corrected_sign = ((inv_ss->unpacked_data()[word_idx + j] % 4 + 4) % 4 >> 1);
                    assert(corrected_sign >= 0 && corrected_sign <= 1);
                    signs_word |= (corrected_sign << j);
                }
                packed_signs[w] = signs_word;
            }
            if (!global_tx) {
                xs->flag_colmajor();
                zs->flag_colmajor();
            }
        }

        for_parallel_y(w, num_qubits) {
            const word_std_t qubit_index_per_word = (w & WORD_MASK);
            for_parallel_x(q, num_words_major) {
                word_std_t inv_word_x = 0;
                word_std_t inv_word_z = 0;
                const size_t block_idx = q * WORD_BITS * num_words_minor + WORD_OFFSET(w);
                #pragma unroll
                for (uint32 k = 0; k < WORD_BITS; k++) {
                    const size_t src_word_idx = k * num_words_minor + block_idx;
                    const word_std_t qubits_word_x = (*inv_xs)[src_word_idx];
                    const word_std_t qubits_word_z = (*inv_zs)[src_word_idx];
                    const word_std_t qubit_bit_x = (qubits_word_x >> qubit_index_per_word) & 1;
                    const word_std_t qubit_bit_z = (qubits_word_z >> qubit_index_per_word) & 1;
                    inv_word_x |= (qubit_bit_x << k);
                    inv_word_z |= (qubit_bit_z << k);
                }
                const size_t dest_word_idx = q + w * num_words_major;
                (*xs)[dest_word_idx] = inv_word_x;
                (*zs)[dest_word_idx] = inv_word_z;
            }
        }
    }

    void Simulator::transpose(const bool& row_major, const hipStream_t& stream) {
        const size_t num_words_minor = inv_tableau.num_words_minor();
        const size_t num_words_major = inv_tableau.num_words_major();
        dim3 currentblock, currentgrid;

        if (row_major) {

            print_tableau(tableau, -1, false);

            // if (options.tune_transpose2r) {
            //     SYNCALL;
            //     tune_transpose(transpose_to_rowmajor, "Transposing to row-major", 
            //     bestblocktranspose2r, bestgridtranspose2r, 
            //     0, false,        // shared size, extend?
            //     num_words_major, // x-dim
            //     2 * num_qubits,  // y-dim 
            //     XZ_TABLE(inv_tableau), inv_tableau.signs(), XZ_TABLE(tableau), tableau.signs(), num_words_major, num_words_minor, num_qubits);
            // }
            // TRIM_BLOCK_IN_DEBUG_MODE(bestblocktranspose2r, bestgridtranspose2r, num_words_major, 2 * num_qubits);
            // currentblock = bestblocktranspose2r, currentgrid = bestgridtranspose2r;
            // TRIM_GRID_IN_XY(num_words_major, 2 * num_qubits);
            // transpose_to_rowmajor <<< currentgrid, currentblock, 0, stream >>> (XZ_TABLE(inv_tableau), inv_tableau.signs(), XZ_TABLE(tableau), tableau.signs(), num_words_major, num_words_minor, num_qubits);
            // if (options.sync) {
            //     LASTERR("failed to launch transpose_to_rowmajor kernel");
            //     SYNC(stream);
            // }

            //print_tableau(inv_tableau, -1, false);

            hipEvent_t start, stop;
            float elapsedTime;

            hipEventCreate(&start);
            hipEventCreate(&stop);

            CHECK(hipEventRecord(start, 0));

            dim3 threadsPerBlock_transpose(WORD_BITS, 1);
            dim3 blocksPerGrid_transpose(num_words_minor, num_words_minor);

            transpose_kernel << <blocksPerGrid_transpose, threadsPerBlock_transpose >> > (XZ_TABLE(tableau), num_words_major, num_words_minor);
            //transpose_kernel << <1, 1 >> > (XZ_TABLE(tableau), num_words_major, num_words_minor);
            LASTERR("transpose failed");
            CHECK(hipDeviceSynchronize());

            printf("after transpose:\n");
            print_tableau(tableau, -1, false);

            dim3 threadsPerBlock_swap(WORD_BITS, 1);
            dim3 blocksPerGrid_swap(num_words_minor, num_words_minor);

            swap_kernel << <blocksPerGrid_swap, threadsPerBlock_swap >> > (tableau.zdata(), num_words_major, num_words_minor);
            //swap_kernel << <1, 1 >> > (tableau.zdata(), num_words_major, num_words_minor);
            LASTERR("swap failed");

            CHECK(hipEventRecord(stop, 0));
            CHECK(hipEventSynchronize(stop));
            CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
            printf("GPU Transpose Time: %f ms\n", elapsedTime);

            hipEventDestroy(start);
            hipEventDestroy(stop);

            printf("after swap:\n");
            print_tableau(tableau, -1, false);

            //hipDeviceReset();
            exit(0);
        }
        else {
            if (options.tune_transpose2c) {
                SYNCALL;
                tune_transpose(transpose_to_colmajor, "Transposing to column-major", 
                bestblocktranspose2c, bestgridtranspose2c, 
                0, false,        // shared size, extend?
                num_words_major, // x-dim
                num_qubits,      // y-dim 
                XZ_TABLE(tableau), tableau.signs(), XZ_TABLE(inv_tableau), inv_tableau.signs(), num_words_major, num_words_minor, num_qubits);
            }
            TRIM_BLOCK_IN_DEBUG_MODE(bestblocktranspose2c, bestgridtranspose2c, num_words_major, num_qubits);
            currentblock = bestblocktranspose2c, currentgrid = bestgridtranspose2c;     
            TRIM_GRID_IN_XY(num_words_major, num_qubits);
            transpose_to_colmajor <<< currentgrid, currentblock, 0, stream >>> (XZ_TABLE(tableau), tableau.signs(), XZ_TABLE(inv_tableau), inv_tableau.signs(), num_words_major, num_words_minor, num_qubits);
            if (options.sync) {
                LASTERR("failed to launch transpose_to_colmajor kernel");
                SYNC(stream);
            }
        }
    }

}

