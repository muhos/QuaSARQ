#include "simulator.hpp"
#include "injectswap.cuh"

namespace QuaSARQ {

    void Simulator::measure(const size_t& p, const depth_t& depth_level, const bool& reversed) {
        assert(options.streams >= 4);
        hipStream_t copy_stream1 = copy_streams[0];
        hipStream_t copy_stream2 = copy_streams[1];
        hipStream_t kernel_stream1 = kernel_streams[0];
        hipStream_t kernel_stream2 = kernel_streams[1];

        const size_t num_words_minor = tableau.num_words_minor();
        const size_t num_words_major = tableau.num_words_major();
        const size_t num_gates_per_window = circuit[depth_level].size();

        reset_pivots(num_gates_per_window, kernel_stream2);

        transpose(true, kernel_stream1);

        // Sync copying gates to device.
        SYNC(copy_stream1);
        SYNC(copy_stream2);
        // Sync resetting pivots.
        SYNC(kernel_stream2);

        find_pivots(num_gates_per_window, kernel_stream1);

        // Copy source pivots to host.
        pivoting.copypivots(kernel_stream1, num_gates_per_window);

        // Reset pivots on device side.
        reset_pivots(num_gates_per_window, kernel_stream1);

        // Sync pivots wth host.
        SYNC(kernel_stream1);

        if (options.tune_measurement)
            tune_assuming_maximum_targets(depth_level);
        else {
            int64 random_measures = measure_indeterminate(depth_level, kernel_stream1);
            stats.circuit.measure_stats.random += random_measures;
            stats.circuit.measure_stats.definite += num_gates_per_window - random_measures;
            stats.circuit.measure_stats.random_per_window = MAX(random_measures, stats.circuit.measure_stats.random_per_window);
        }

        transpose(false, kernel_stream1);
    }

    void Simulator::tune_assuming_maximum_targets(const depth_t& depth_level) {
		const size_t num_words_minor = tableau.num_words_minor();
        const size_t num_words_major = tableau.num_words_major();
        const size_t num_qubits_padded = tableau.num_qubits_padded();
        const size_t num_gates_per_window = circuit[depth_level].size();
		uint32 max_targets = ROUNDUP(num_qubits, 10);
        pivot_t min_pivot = 0;
        
        LOG2(2, "Tuning measurements for maximum targets of %u for pivot %u", max_targets, min_pivot);

        Vec<pivot_t> h_pivots(max_targets + 1);
        h_pivots[0] = min_pivot;

        for (uint32 i = 0; i < max_targets; i++) {
            h_pivots[i + 1] = rand() % num_qubits;
        }

        CHECK(hipMemcpy(pivoting.pivots, h_pivots.data(), sizeof(pivot_t) * (max_targets + 1), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(pivoting.d_active_pivots, &max_targets, sizeof(uint32), hipMemcpyHostToDevice));

        prefix.tune_inject_cx(tableau, pivoting.pivots, max_targets);
        if (options.tune_injectswap) {
            SYNCALL;
            tune_inject_swap(inject_swap_k,
                        bestblockinjectswap, 
                        bestgridinjectswap, 
                        XZ_TABLE(tableau),
                        tableau.signs(),
                        pivoting.pivots,
                        num_words_major,
                        num_words_minor,
                        num_qubits_padded);
        }
	}

    int64 Simulator::measure_indeterminate(const depth_t& depth_level, const hipStream_t& stream) {
        const size_t num_words_minor = tableau.num_words_minor();
        const size_t num_words_major = tableau.num_words_major();
        const size_t num_gates_per_window = circuit[depth_level].size();
        const size_t num_qubits_padded = tableau.num_qubits_padded();
        pivot_t* host_pivots = pivoting.host_pivots;
        if (options.check_measurement) {
            mchecker.copy_input(tableau);
            mchecker.check_initial_pivots(circuit, depth_level, host_pivots, num_gates_per_window);
        }
        int64 random_measures = 0;
        for(size_t i = 0; i < num_gates_per_window; i++) {
            const Gate& curr_gate = circuit.gate(depth_level, i);
            const pivot_t curr_pivot = host_pivots[i];
            const qubit_t qubit = curr_gate.wires[0];
            if (curr_pivot != INVALID_PIVOT) {
                compact_targets(qubit, stream);
                SYNC(stream);
                const uint32 active_pivots = pivoting.h_active_pivots[0];
                if (options.check_measurement)
                    mchecker.check_compact_pivots(qubit, pivoting.pivots, active_pivots);
                if (active_pivots) {
                    if (active_pivots > 1)
                        inject_cx(active_pivots - 1/*active_pivots - 1*/, stream);
                    inject_swap(qubit, stream);
                    random_measures++;
                }
            }
        }

        return random_measures;
    }
}

