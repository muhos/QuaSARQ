#include "simulator.hpp"
#include "injectx.cuh"
#include "injectswap.cuh"

namespace QuaSARQ {

    void Simulator::measure(const size_t& p, const depth_t& depth_level, const bool& reversed) {
        assert(options.streams >= 4);
        hipStream_t copy_stream1 = copy_streams[0];
        hipStream_t copy_stream2 = copy_streams[1];
        hipStream_t kernel_stream1 = kernel_streams[0];
        hipStream_t kernel_stream2 = kernel_streams[1];

        const size_t num_words_minor = tableau.num_words_minor();
        const size_t num_words_major = tableau.num_words_major();
        const size_t num_gates_per_window = circuit[depth_level].size();

        reset_pivots(num_gates_per_window, kernel_stream2);

        transpose(true, kernel_stream1);

        // Sync copying gates to device.
        SYNC(copy_stream1);
        SYNC(copy_stream2);
        // Sync resetting pivots.
        SYNC(kernel_stream2);

        find_pivots(num_gates_per_window, kernel_stream1);

        // Copy source pivots to host.
        pivoting.copypivots(kernel_stream1, num_gates_per_window);

        // Reset pivots on device side.
        reset_pivots(num_gates_per_window, kernel_stream1);

        // Sync pivots wth host.
        SYNC(kernel_stream1);

        if (options.tune_measurement)
            tune_assuming_maximum_targets(depth_level);
        else {
            int64 random_measures = measure_indeterminate(depth_level, kernel_stream1);
            stats.circuit.measure_stats.random += random_measures;
            stats.circuit.measure_stats.definite += num_gates_per_window - random_measures;
            stats.circuit.measure_stats.random_per_window = MAX(random_measures, stats.circuit.measure_stats.random_per_window);
        }

        transpose(false, kernel_stream1);
    }

    void Simulator::tune_assuming_maximum_targets(const depth_t& depth_level) {
		const size_t num_words_minor = tableau.num_words_minor();
        const size_t num_words_major = tableau.num_words_major();
        const size_t num_qubits_padded = tableau.num_qubits_padded();
        const size_t num_gates_per_window = circuit[depth_level].size();
		uint32 max_targets = MAX(10, ROUNDUP(num_qubits, 10));
        const pivot_t min_pivot = 0;
        const qubit_t qubit = 0;
        
        LOG2(2, "Tuning measurements for maximum targets of %u for pivot %u", max_targets, min_pivot);

        if (options.tune_newpivots) {
            SYNCALL;
            tune_finding_new_pivots(anti_commuting_pivots, 
                bestblocknewpivots, bestgridnewpivots, 
                sizeof(pivot_t),
                pivoting.pivots, 
                tableau.xtable(), 
                qubit, 
                num_qubits, 
                num_words_major, 
                num_words_minor,
                num_qubits_padded);
            reset_all_pivots <<<bestgridreset, bestblockreset>>> (pivoting.pivots, num_qubits);
            SYNCALL;
        }

        Vec<pivot_t> h_pivots(max_targets + 1);
        h_pivots[0] = min_pivot;
        for (uint32 i = 0; i < max_targets; i++) {
            h_pivots[i + 1] = rand() % num_qubits;
        }

        CHECK(hipMemcpy(pivoting.pivots, h_pivots.data(), sizeof(pivot_t) * (max_targets + 1), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(pivoting.d_active_pivots, &max_targets, sizeof(uint32), hipMemcpyHostToDevice));
        
        prefix.tune_inject_cx(tableau, pivoting.pivots, max_targets);

        assert(SIGN_FLAG_IDX > COMMUTING_FLAG_IDX);
        const size_t num_copies = SIGN_FLAG_IDX + 1; 
        h_pivots[COMMUTING_FLAG_IDX] = 1; // Assume commutation is true.
        h_pivots[SIGN_FLAG_IDX] = 1; // Enable injecting x-gate.
        CHECK(hipMemcpy(pivoting.pivots, h_pivots.data(), sizeof(pivot_t) * num_copies, hipMemcpyHostToDevice));
        
        if (options.tune_injectswap) {
            SYNCALL;
            tune_inject_swap(inject_swap_k,
                        bestblockinjectswap, 
                        bestgridinjectswap, 
                        XZ_TABLE(tableau),
                        tableau.signs(),
                        pivoting.pivots,
                        qubit,
                        1,
                        num_words_major,
                        num_words_minor,
                        num_qubits_padded);
        }

        if (options.tune_injectx) {
            SYNCALL;
            tune_inject_x(inject_x_k,
                        bestblockinjectx, 
                        bestgridinjectx, 
                        XZ_TABLE(tableau),
                        tableau.signs(),
                        pivoting.pivots,
                        num_words_major,
                        num_words_minor,
                        num_qubits_padded);
        }
	}

    int64 Simulator::measure_indeterminate(const depth_t& depth_level, const hipStream_t& stream) {
        const size_t num_words_minor = tableau.num_words_minor();
        const size_t num_words_major = tableau.num_words_major();
        const size_t num_gates_per_window = circuit[depth_level].size();
        const size_t num_qubits_padded = tableau.num_qubits_padded();
        pivot_t* host_pivots = pivoting.host_pivots;
        if (options.check_measurement) {
            mchecker.copy_input(tableau);
            mchecker.check_initial_pivots(circuit, depth_level, host_pivots, num_gates_per_window);
        }
        int64 random_measures = 0;
        for(size_t i = 0; i < num_gates_per_window; i++) {
            const Gate& curr_gate = circuit.gate(depth_level, i);
            const pivot_t curr_pivot = host_pivots[i];
            const qubit_t qubit = curr_gate.wires[0];
            if (curr_pivot != INVALID_PIVOT) {
                compact_targets(qubit, stream);
                SYNC(stream);
                const uint32 active_pivots = pivoting.h_active_pivots[0];
                if (options.check_measurement)
                    mchecker.check_compact_pivots(qubit, pivoting.pivots, active_pivots);
                if (active_pivots) {
                    if (active_pivots > 1) {
                        inject_cx(active_pivots - 1, stream);
                    }
                    const sign_t rbit = mrand.brand();
                    inject_swap(qubit, rbit, stream);
                    inject_x(qubit, rbit, stream);
                    random_measures++;
                }
            }
        }

        return random_measures;
    }
}

