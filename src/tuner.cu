#include "hip/hip_runtime.h"
#include "simulator.hpp"
#include "tuner.cuh"

namespace QuaSARQ {

	// Even number so that changes can be undone at the last sample.
	constexpr size_t NSAMPLES = 2;
	constexpr size_t TRIALS = size_t(1e3);
	constexpr double PRECISION = 0.001;
	constexpr int MIN_PRECISION_HITS = 2;
#if	defined(_DEBUG) || defined(DEBUG) || !defined(NDEBUG)
	constexpr grid_t maxThreadsPerBlock = 512;
#else
	constexpr grid_t maxThreadsPerBlock = 1024;
#endif
	constexpr grid_t maxThreadsPerBlockY = 512;
	constexpr grid_t maxThreadsPerBlockX = 256;
	constexpr grid_t initThreadsPerBlock = 2;

	Tuner::Tuner() : 
		Simulator() 
	{ }

	void Tuner::write() {
		string config = std::to_string(num_qubits) + " ";
		if (options.tune_identity) {
			config += "I " + std::to_string(bestGridIdentity.x) + " " + std::to_string(bestBlockIdentity.x) + " ";
		}
		if (options.tune_step) {
			config += "S " + std::to_string(bestGridStep.x) + " " + std::to_string(bestGridStep.y) + " ";
			config += std::to_string(bestBlockStep.x) + " " + std::to_string(bestBlockStep.y) + " ";
		}
		config += "\n";
		fwrite(config.c_str(), 1, config.size(), configfile);
	}

	void Tuner::reset() {
		// Force kernel tuner to run on all kernels.
		bestBlockIdentity = dim3();
		bestGridIdentity = dim3();
		bestBlockReset = dim3();
		bestGridReset = dim3();
		bestBlockStep = dim3();
		bestGridStep = dim3();
	}

	void Tuner::run() {
		if (!open_config("w"))
			LOGERROR("cannot tune without opening a configuration file");
		// Create a tableau in GPU memory for the maximum qubits.
		const size_t max_num_qubits = num_qubits;
		num_partitions = tableau.alloc(max_num_qubits, max_window_bytes, measuring);
		if (num_partitions > 1) num_partitions = 1;
		gpu_circuit.initiate(max_parallel_gates, max_parallel_gates_buckets);
		// Start tuning simulation with max qubits.
		do {
			LOGN2(1, "Tuning all kernels for %s%zd qubits%s, %zd partition..", CREPORTVAL, num_qubits, CNORMAL, num_partitions);
			// Reset old configuration.
			reset();
			// Tune identity.
			identity(tableau, 0, tableau.num_words_per_column() * WORD_BITS, custreams, options.initialstate);
			// Parse a circuit.
			parse();
			// Start step-wise simulation.
			simulate(0, false);
			// Write configurations.
			write();
			// Clean old circuit.
			circuit.destroy();
			// Decrease qubits.
			num_qubits = num_qubits >= options.tuner_step_qubits ? num_qubits - options.tuner_step_qubits : 0;

		} while (num_qubits >= options.tuner_initial_qubits);
		close_config();
	}

	// Benchmark a 'kernel' up to NSAMPLES times and record the time
	// in AVGTIME per ms. Variables grid and block are assumed.
#define BENCHMARK_KERNEL(AVGTIME, NSAMPLES, SHAREDSIZE, ...) \
	do { \
		double runtime = 0; \
		for (size_t sample = 0; sample < NSAMPLES; sample++) { \
			cutimer.start(); \
			kernel <<< grid, block, SHAREDSIZE >>> ( __VA_ARGS__ ); \
			LASTERR("failed to launch kernel for benchmarking"); \
			cutimer.stop(); \
			runtime += cutimer.time(); \
		} \
		AVGTIME = (runtime / NSAMPLES); \
	} while(0)

	// Given TIME and MIN, update BESTGRID and BESTBLOCK.
	// Assume block and grid are defined.
#define BEST_CONFIG(TIME, MIN, BESTGRID, BESTBLOCK, BAILOUT) \
	if (TIME < MIN) { \
		if ((MIN - TIME) <= PRECISION && !--min_precision_hits) { \
			LOG2(1, " Found slightly better GPU Time for block(x:%u, y:%u) and grid(x:%u, y:%u): %f ms", block.x, block.y, grid.x, grid.y, TIME); \
			BAILOUT = true; \
		} \
		MIN = TIME; \
		BESTBLOCK = block; \
		BESTGRID = grid; \
		if (!BAILOUT) LOG2(1, " Found better GPU Time for block(x:%u, y:%u) and grid(x:%u, y:%u): %f ms", block.x, block.y, grid.x, grid.y, TIME); \
	}

#define TUNE_1D(...) \
	{ \
		if (bestBlock.x > 1 || bestGrid.x > 1) { \
			LOG2(2, "\nBest configuration: block(%d), grid(%d) will be used without tuning.", bestBlock.x, bestGrid.x); \
			return; \
		} \
		LOG0(""); \
		LOG2(1, "Tunning %s kernel with maximum of %zd trials and %-.5f milliseconds precision...", opname, TRIALS, PRECISION); \
		int min_precision_hits = MIN_PRECISION_HITS;  \
		const grid_t maxBlocksPerGrid = maxGPUBlocks << 1; \
		OPTIMIZEBLOCKS(initBlocksPerGrid, size, initThreadsPerBlock); \
		double minRuntime = double(UINTMAX_MAX); \
		bool early_exit = false; \
		size_t trials = 0; \
		for (grid_t gridX = initBlocksPerGrid; gridX <= maxBlocksPerGrid && !early_exit && trials < TRIALS; gridX += 4, trials++) { \
			for (grid_t blockX = initThreadsPerBlock; blockX <= maxThreadsPerBlock && !early_exit && trials < TRIALS; blockX <<= 1, trials++) { \
				if (blockX > maxWarpSize && blockX % maxWarpSize != 0) \
					continue; \
				dim3 block((uint32)blockX); \
				dim3 grid((uint32)gridX); \
				double avgRuntime = 0; \
				BENCHMARK_KERNEL(avgRuntime, NSAMPLES, 0, ## __VA_ARGS__); \
				BEST_CONFIG(avgRuntime, minRuntime, bestGrid, bestBlock, early_exit); \
			} \
		} \
		LOG0(""); \
		LOG2(1, "Best GPU time for %s operation using block(%d, 1), and grid(%d, 1): %f ms", opname, bestBlock.x, bestGrid.x, minRuntime); \
		LOG0(""); \
		fflush(stdout); \
	}

#define TUNE_2D(...) \
	{ \
		if (bestBlock.x > 1 || bestGrid.x > 1 || bestBlock.y > 1 || bestGrid.y > 1) { \
			LOG2(2, "\nBest configuration: block(%d, %d), grid(%d, %d) will be used without tuning.", bestBlock.x, bestBlock.y, bestGrid.x, bestGrid.y); \
			return; \
		} \
		LOG0(""); \
		LOG2(1, "Tunning %s kernel with maximum of %zd trials and %-.5f milliseconds precision...", opname, TRIALS, PRECISION); \
		int min_precision_hits = MIN_PRECISION_HITS; \
		const bool x_warped = hasstr(opname, "warped"); \
		OPTIMIZEBLOCKS2D(initBlocksPerGridY, data_size_in_y, initThreadsPerBlock); \
		OPTIMIZEBLOCKS2D(initBlocksPerGridX, data_size_in_x, initThreadsPerBlock); \
		double minRuntime = double(UINTMAX_MAX); \
		bool early_exit = false; \
		size_t trials = 0; \
		initBlocksPerGridY = (grid_t) ceil(initBlocksPerGridY / 1.0); \
		initBlocksPerGridX = (grid_t) ceil(initBlocksPerGridX / 1.0); \
		const grid_t maxBlocksPerGridY = maxGPUBlocks2D; \
		const grid_t maxBlocksPerGridX = maxGPUBlocks2D << 1; \
		for (grid_t blocksY = initBlocksPerGridY; (blocksY <= maxBlocksPerGridY) && !early_exit && trials < TRIALS; blocksY += 8, trials++) { \
			for (grid_t blocksX = initBlocksPerGridX; (blocksX <= maxBlocksPerGridX) && !early_exit && trials < TRIALS; blocksX += 8, trials++) { \
				for (grid_t threadsY = initThreadsPerBlock; (threadsY <= maxThreadsPerBlockY) && !early_exit && trials < TRIALS; threadsY <<= 1) { \
					for (grid_t threadsX = initThreadsPerBlock; (threadsX <= maxThreadsPerBlockX) && !early_exit && trials < TRIALS; threadsX <<= 1) { \
						const grid_t threadsPerBlock = threadsX * threadsY; \
						const size_t extended_shared_size = shared_size_yextend ? shared_element_bytes * threadsPerBlock : shared_element_bytes * threadsX; \
						if (x_warped && threadsX > maxWarpSize) continue; \
						if (extended_shared_size > maxGPUSharedMem || threadsPerBlock > maxThreadsPerBlock) continue; \
						/* Avoid deadloack due to warp divergence. */ \
						if ((threadsX > maxWarpSize && threadsX % maxWarpSize != 0) || (threadsY > maxWarpSize && threadsY % maxWarpSize != 0)) \
							continue; \
						dim3 block((uint32)threadsX, (uint32)threadsY); \
						dim3 grid((uint32)blocksX, (uint32)blocksY); \
						double avgRuntime = 0; \
						BENCHMARK_KERNEL(avgRuntime, NSAMPLES, extended_shared_size, ## __VA_ARGS__); \
						BEST_CONFIG(avgRuntime, minRuntime, bestGrid, bestBlock, early_exit); \
					} \
				} \
			} \
		} \
		LOG2(1, "Best %s configuration found after %zd trials:", opname, trials); \
		LOG2(1, " Block (%-4u, %4u)", bestBlock.x, bestBlock.y); \
		LOG2(1, " Grid  (%-4u, %4u)", bestGrid.x, bestGrid.y); \
		LOG2(1, " Min time: %.4f ms", minRuntime); \
		LOG0(""); \
		fflush(stdout); \
	}

	void tune_kernel(void (*kernel)(const size_t, const size_t, Table*),
		const char* opname, dim3& bestBlock, dim3& bestGrid,
		const size_t& offset, const size_t& size, Table* ps)
	{
		TUNE_1D(offset, size, ps);
	}

	#ifdef INTERLEAVE_XZ
	#define TUNE_XZ_TABLES ps
	#else
	#define TUNE_XZ_TABLES xs, zs
	#endif

	void tune_kernel(void (*kernel)(const size_t, 
		#ifdef INTERLEAVE_XZ
		Table*,
		#else
		Table*, Table*, 
		#endif
		Signs *),
		const char* opname, dim3& bestBlock, dim3& bestGrid,
		const size_t& size, 
		#ifdef INTERLEAVE_XZ
		Table* ps, 
		#else
		Table* xs, Table* zs, 
		#endif
		Signs *ss)
	{
		TUNE_1D(size, TUNE_XZ_TABLES, ss);
	}

	void tune_kernel(void (*kernel)(const gate_ref_t*, const bucket_t*, const size_t, const size_t, 
		#ifdef INTERLEAVE_XZ
		Table*,
		#else
		Table*, Table*, 
		#endif
		Signs *),
		const char* opname,
		dim3& bestBlock, dim3& bestGrid, 
		const size_t& shared_element_bytes, 
		const bool& shared_size_yextend,
		const size_t& data_size_in_x, 
		const size_t& data_size_in_y,
		const gate_ref_t* gate_refs, const bucket_t* gate_buckets, const size_t& num_gates, const size_t& num_words_per_column, 
		#ifdef INTERLEAVE_XZ
		Table* ps, 
		#else
		Table* xs, Table* zs, 
		#endif
		Signs *ss)
	{
		assert(gate_ref_t(num_gates) == num_gates);
		assert(gate_ref_t(data_size_in_x) == data_size_in_x);
		TUNE_2D(gate_refs, gate_buckets, num_gates, num_words_per_column, TUNE_XZ_TABLES, ss);
	}

}

