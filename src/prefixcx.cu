#include "hip/hip_runtime.h"

#include "prefix.cuh"
#include "collapse.cuh"
#include "access.cuh"
#include "vector.hpp"
#include "print.cuh"
#include "prefixcxcub.cuh"
#include "prefixdim.cuh"

namespace QuaSARQ {

    __global__ 
    void scan_targets_pass_2(
                Table *         prefix_xs, 
                Table *         prefix_zs, 
                Table *         inv_xs, 
                Table *         inv_zs,
                Signs *         inv_ss,
        const   word_std_t *    block_intermediate_prefix_z,
        const   word_std_t *    block_intermediate_prefix_x,
        const   Commutation *   commutations,
        const   uint32          pivot,
        const   size_t          total_targets,
        const   size_t          num_words_major,
        const   size_t          num_words_minor,
        const   size_t          num_qubits_padded,
        const   size_t          max_blocks,
        const   size_t          pass_1_blocksize)
    { 
        word_std_t *shared = SharedMemory<word_std_t>();
        word_std_t *shared_z = shared;
        word_std_t *shared_x = shared_z + blockDim.x;
        word_std_t *signs_destab = shared_x + blockDim.x;
        word_std_t *signs_stab = signs_destab + blockDim.x;
        grid_t      collapse_tid = threadIdx.y * 4 * blockDim.x + threadIdx.x;
        word_std_t *xs = inv_xs->words();
        word_std_t *zs = inv_zs->words();

        for_parallel_y(w, num_words_minor) {

            const size_t c_destab = TABLEAU_INDEX(w, pivot);
            const size_t c_stab = c_destab + TABLEAU_STAB_OFFSET;

            word_std_t zc_destab = 0;
            word_std_t xc_destab = 0;
            word_std_t xc_and_zt = 0;
            word_std_t not_zc_xor_xt = 0;
            word_std_t local_destab_sign = 0;
            word_std_t local_stab_sign = 0;

            for_parallel_x(tid_x, total_targets) {

                size_t t = tid_x + pivot + 1;

                if (commutations[t].anti_commuting) {

                    const size_t t_destab = TABLEAU_INDEX(w, t);
                    const size_t t_stab = t_destab + TABLEAU_STAB_OFFSET;

                    assert(c_destab < inv_zs->size());
                    assert(t_destab < inv_zs->size());

                    const size_t word_idx = PREFIX_TABLEAU_INDEX(w, tid_x);
                    word_std_t zc_xor_zt = (*prefix_zs)[word_idx];
                    word_std_t xc_xor_xt = (*prefix_xs)[word_idx];

                    // Compute final prefixes and hence final {x,z}'c = {x,z}'c ^ {x,z}'t expressions.
                    const size_t bid = w * max_blocks + (tid_x / pass_1_blocksize);
                    zc_xor_zt ^= block_intermediate_prefix_z[bid];
                    xc_xor_xt ^= block_intermediate_prefix_x[bid];

                    // Compute the CX expression for Z.
                    word_std_t c_stab_word = zs[c_stab];
                    word_std_t t_destab_word = zs[t_destab];
                    xc_and_zt = (c_stab_word & t_destab_word);
                    not_zc_xor_xt = ~(zc_xor_zt ^ zs[t_stab]);
                    local_destab_sign ^= xc_and_zt & not_zc_xor_xt;
                    
                    // Update Z tableau.
                    zs[t_stab] ^= c_stab_word;
                    zc_destab ^= t_destab_word; // requires collapse.

                    // Compute the CX expression for X.
                    c_stab_word = xs[c_stab];
                    t_destab_word = xs[t_destab];
                    xc_and_zt = (c_stab_word & t_destab_word);
                    not_zc_xor_xt = ~(xc_xor_xt ^ xs[t_stab]);
                    local_stab_sign ^= xc_and_zt & not_zc_xor_xt;

                    // Update X tableau.
                    xs[t_stab] ^= c_stab_word;
                    xc_destab ^= t_destab_word; // requires collapse.
                }
            }

            collapse_load_shared_dual(shared_z, zc_destab, shared_x, xc_destab, collapse_tid, total_targets);
            collapse_shared_dual(shared_z, zc_destab, shared_x, xc_destab, collapse_tid);
            collapse_warp_dual(shared_z, zc_destab, shared_x, xc_destab, collapse_tid);
            collapse_load_shared_dual(signs_destab, local_destab_sign, signs_stab, local_stab_sign, collapse_tid, total_targets);
            collapse_shared_dual(signs_destab, local_destab_sign, signs_stab, local_stab_sign, collapse_tid);
            collapse_warp_dual(signs_destab, local_destab_sign, signs_stab, local_stab_sign, collapse_tid);

            if (!threadIdx.x) {
                if (zc_destab)
                    atomicXOR(zs + c_destab, zc_destab);
                if (xc_destab)
                    atomicXOR(xs + c_destab, xc_destab);
                if (local_destab_sign)
                    atomicXOR(inv_ss->data(w), local_destab_sign);
                if (local_stab_sign)
                    atomicXOR(inv_ss->data(w + num_words_minor), local_stab_sign);
            }
        }
    }

    void call_injectcx_pass_1_kernel(
                Tableau<DeviceAllocator>& targets, 
                Tableau<DeviceAllocator>& input,
                word_std_t *        block_intermediate_prefix_z,
                word_std_t *        block_intermediate_prefix_x,
        const   Commutation *       commutations,
        const   uint32              pivot,
        const   size_t              total_targets,
        const   size_t              num_words_major,
        const   size_t              num_words_minor,
        const   size_t              num_qubits_padded,
        const   size_t              max_blocks,
        const   dim3&               currentblock,
        const   dim3&               currentgrid,
        const   hipStream_t&       stream) {
        
        GENERATE_SWITCH_FOR_CALL(CALL_INJECTCX_PASS_1_FOR_BLOCK)
    }

    // We need to compute prefix-xor of t-th destabilizer in X,Z for t = c+1, c+2, ... c+n-1
    // so that later we can xor every prefix-xor with controlled destabilizer.
    void Prefix::inject_CX(Tableau<DeviceAllocator>& input, const Commutation* commutations, const uint32& pivot, const qubit_t& qubit, const hipStream_t& stream) {
        assert(num_qubits > pivot);
        assert(nextPow2(MIN_BLOCK_INTERMEDIATE_SIZE) == MIN_BLOCK_INTERMEDIATE_SIZE);
        
        const size_t num_qubits_padded = input.num_qubits_padded();

        // Calculate number of target generators.
        const size_t total_targets = num_qubits - pivot - 1;
        if (!total_targets) return;

        // Do the first phase of prefix.
        dim3 currentblock, currentgrid;
        if (options.tune_injectprepare) {
            SYNCALL;
            tune_inject_pass_1(
                bestblockinjectprepare, bestgridinjectprepare,
                2 * sizeof(word_std_t), // used to skip very large blocks.
                total_targets,
                num_words_minor,
                targets, 
                input, 
                zblocks(), 
                xblocks(),
                commutations, 
                pivot,
                total_targets, 
                num_words_major, 
                num_words_minor,
                num_qubits_padded,
                max_intermediate_blocks
            );
            SYNCALL;
        }
        TRIM_Y_BLOCK_IN_DEBUG_MODE(bestblockinjectprepare, bestgridinjectprepare, num_words_minor);
        currentblock = bestblockinjectprepare, currentgrid = bestgridinjectprepare;
        TRIM_GRID_IN_XY(total_targets, num_words_minor);
        const size_t pass_1_blocksize = currentblock.x;
        const size_t pass_1_gridsize = ROUNDUP(total_targets, pass_1_blocksize);
        if (pass_1_gridsize > max_intermediate_blocks)
            LOGERROR("too many blocks for intermediate arrays.");
        LOGN2(2, " Running pass-1 kernel with block(x:%u, y:%u) and grid(x:%u, y:%u).. ",
            currentblock.x, currentblock.y, currentgrid.x, currentgrid.y);
        if (options.sync) cutimer.start(stream);
        call_injectcx_pass_1_kernel(
            targets, 
            input, 
            zblocks(), 
            xblocks(),
            commutations, 
            pivot,
            total_targets, 
            num_words_major, 
            num_words_minor,
            num_qubits_padded,
            max_intermediate_blocks,
            currentblock,
            currentgrid,
            stream
        );
        if (options.sync) {
            LASTERR("failed to scan targets in pass 1");
            cutimer.stop(stream);
            LOGENDING(2, 4, "(time %.3f ms)", cutimer.time());
        } else LOGDONE(2, 4);

        // Verify pass-1 prefix.
        assert(checker.check_prefix_pass_1(
            targets,
            input,
            commutations,
            zblocks(), 
            xblocks(),
            qubit,
            pivot,
            total_targets,
            num_words_major,
            num_words_minor,
            num_qubits_padded,
            max_intermediate_blocks,
            pass_1_blocksize,
            pass_1_gridsize
        ));

        // Intermeditae scan of blocks resulted in pass 1.
        scan_blocks(nextPow2(pass_1_gridsize), pass_1_blocksize, stream);

        // Verify intermediate-pass prefix.
        assert(checker.check_prefix_intermediate_pass(
            zblocks(), 
            xblocks(),
            qubit,
            pivot,
            num_words_minor,
            max_intermediate_blocks,
            pass_1_gridsize
        ));

        // Second phase of injecting CX.
        if (options.tune_injectfinal) {
            SYNCALL;
            tune_inject_pass_2(
                scan_targets_pass_2, 
                bestblockinjectfinal, bestgridinjectfinal,
                4 * sizeof(word_std_t),
                total_targets,
                num_words_minor,
                XZ_TABLE(targets), 
                XZ_TABLE(input),
                input.signs(),
                zblocks(), 
                xblocks(), 
                commutations, 
                pivot, 
                total_targets, 
                num_words_major, 
                num_words_minor, 
                num_qubits_padded,
                max_intermediate_blocks,
                pass_1_blocksize
            );
            SYNCALL;
        }
        TRIM_BLOCK_IN_DEBUG_MODE(bestblockinjectfinal, bestgridinjectfinal, total_targets, num_words_minor);
        currentblock = bestblockinjectfinal, currentgrid = bestgridinjectfinal;
        TRIM_GRID_IN_XY(total_targets, num_words_minor);
        OPTIMIZESHARED(finalize_prefix_smem_size, currentblock.y * currentblock.x, 4 * sizeof(word_std_t));
        LOGN2(2, " Running pass-2 kernel with block(x:%u, y:%u) and grid(x:%u, y:%u).. ", \
            currentblock.x, currentblock.y, currentgrid.x, currentgrid.y); \
        if (options.sync) cutimer.start(stream);
        scan_targets_pass_2 <<<currentgrid, currentblock, finalize_prefix_smem_size, stream>>> (
            XZ_TABLE(targets), 
            XZ_TABLE(input),
            input.signs(),
            zblocks(), 
            xblocks(), 
            commutations, 
            pivot, 
            total_targets, 
            num_words_major, 
            num_words_minor, 
            num_qubits_padded,
            max_intermediate_blocks,
            pass_1_blocksize
        );
        if (options.sync) {
            LASTERR("failed to scan targets in pass 2");
            cutimer.stop(stream);
            LOGENDING(2, 4, "(time %.3f ms)", cutimer.time());
        } else LOGDONE(2, 4);

        // Verify pass-2 prefix.
        assert(checker.check_prefix_pass_2(
            targets, 
            input,
            qubit,
            pivot, 
            total_targets, 
            num_words_major, 
            num_words_minor, 
            num_qubits_padded,
            max_intermediate_blocks,
            pass_1_blocksize
        ));
    }

}