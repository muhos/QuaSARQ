#include "hip/hip_runtime.h"

#include "prefix.cuh"
#include "collapse.cuh"
#include "access.cuh"
#include "vector.hpp"
#include "print.cuh"
#include "templatedim.cuh"
#include "datatypes.cuh"
#include "warp.cuh"
#include <cub/block/block_scan.cuh>
#include <cub/block/block_reduce.cuh>

namespace QuaSARQ {


    template <int BLOCKX, int BLOCKY>
    __global__ 
    void scan_targets_pass_1(
                Table *             prefix_xs, 
                Table *             prefix_zs, 
                word_std_t *        block_intermediate_prefix_z,
                word_std_t *        block_intermediate_prefix_x,
                ConstTablePointer   inv_xs, 
                ConstTablePointer   inv_zs,
        const   pivot_t*            pivots,
        const   size_t              active_targets,
        const   size_t              num_words_major,
        const   size_t              num_words_minor,
        const   size_t              num_qubits_padded,
        const   size_t              max_blocks) {

        assert(active_targets > 0);

        typedef hipcub::BlockScan<word_std_t, BLOCKX, hipcub::BLOCK_SCAN_RAKING> BlockScan;

        __shared__ typename BlockScan::TempStorage temp_storage_z[BLOCKY];
        __shared__ typename BlockScan::TempStorage temp_storage_x[BLOCKY];

        for_parallel_y_tiled(by, num_words_minor) {
            const grid_t w = threadIdx.y + by * blockDim.y;

            for_parallel_x_tiled(bx, active_targets) {
                const grid_t tid_x = threadIdx.x + bx * blockDim.x;
                
                word_std_t z = 0;
                word_std_t x = 0;
                word_std_t init_z = 0;
                word_std_t init_x = 0;

                if (w < num_words_minor && tid_x < active_targets) {
                    const pivot_t pivot = pivots[0];
                    assert(pivot != INVALID_PIVOT);
                    const size_t t = pivots[tid_x + 1];
                    assert(t != pivot);
                    assert(t != INVALID_PIVOT);
                    const size_t t_destab = TABLEAU_INDEX(w, t);
                    z = (*inv_zs)[t_destab];
                    x = (*inv_xs)[t_destab];
                    const size_t c_destab = TABLEAU_INDEX(w, pivot);
                    init_z = (*inv_zs)[c_destab];
                    init_x = (*inv_xs)[c_destab];
                }

                word_std_t blockSum_z;
                word_std_t blockSum_x;

                BlockScan(temp_storage_z[threadIdx.y]).ExclusiveScan(z, z, 0, XOROP(), blockSum_z);
                BlockScan(temp_storage_x[threadIdx.y]).ExclusiveScan(x, x, 0, XOROP(), blockSum_x);

                if (w < num_words_minor && tid_x < active_targets) {
                    const size_t word_idx = PREFIX_TABLEAU_INDEX(w, tid_x);
                    assert(word_idx < prefix_zs->size());
                    assert(word_idx < prefix_xs->size());
                    (*prefix_zs)[word_idx] = init_z ^ z;
                    (*prefix_xs)[word_idx] = init_x ^ x;
                }

                if (w < num_words_minor && threadIdx.x == blockDim.x - 1) {
                    assert((blockIdx.x * num_words_minor + w) < gridDim.x * num_words_minor);
                    const size_t bid = PREFIX_INTERMEDIATE_INDEX(w, bx);
                    block_intermediate_prefix_z[bid] = blockSum_z;
                    block_intermediate_prefix_x[bid] = blockSum_x;
                }
            }
        }
    }

    #define CALL_INJECTCX_PASS_1_FOR_BLOCK(X, Y) \
        scan_targets_pass_1 <X, Y> \
        <<<currentgrid, currentblock, 0, stream>>> ( \
                XZ_TABLE(targets), \
                block_intermediate_prefix_z, \
                block_intermediate_prefix_x, \
                XZ_TABLE(input), \
                pivots, \
                active_targets, \
                num_words_major, \
                num_words_minor, \
                num_qubits_padded, \
                max_blocks \
            )

    template <int BLOCKX, int BLOCKY>
    __global__ 
    void scan_targets_pass_2(
                Table *             inv_xs, 
                Table *             inv_zs,
                Signs *             inv_ss,
                ConstTablePointer   prefix_xs, 
                ConstTablePointer   prefix_zs, 
                ConstWordsPointer   block_intermediate_prefix_z,
                ConstWordsPointer   block_intermediate_prefix_x,
        const   pivot_t*            pivots,
        const   size_t              active_targets,
        const   size_t              num_words_major,
        const   size_t              num_words_minor,
        const   size_t              num_qubits_padded,
        const   size_t              max_blocks,
        const   size_t              pass_1_blocksize)
    { 
        word_std_t *xs = inv_xs->words();
        word_std_t *zs = inv_zs->words();

        for_parallel_y(w, num_words_minor) {

            word_std_t zc_destab = 0;
            word_std_t xc_destab = 0;
            word_std_t xc_and_zt = 0;
            word_std_t not_zc_xor_xt = 0;
            word_std_t local_destab_sign = 0;
            word_std_t local_stab_sign = 0;

            for_parallel_x(tid_x, active_targets) {
                const pivot_t pivot = pivots[0];
                assert(pivot != INVALID_PIVOT);
                const size_t t = pivots[tid_x + 1];
                assert(t != pivot);
                assert(t != INVALID_PIVOT);

                const size_t c_destab = TABLEAU_INDEX(w, pivot);
                const size_t c_stab = c_destab + TABLEAU_STAB_OFFSET;
                const size_t t_destab = TABLEAU_INDEX(w, t);
                const size_t t_stab = t_destab + TABLEAU_STAB_OFFSET;

                assert(c_destab < inv_zs->size());
                assert(t_destab < inv_zs->size());

                const size_t word_idx = PREFIX_TABLEAU_INDEX(w, tid_x);
                word_std_t zc_xor_zt = (*prefix_zs)[word_idx];
                word_std_t xc_xor_xt = (*prefix_xs)[word_idx];

                // Compute final prefixes and hence final {x,z}'c = {x,z}'c ^ {x,z}'t expressions.
                const size_t bid = PREFIX_INTERMEDIATE_INDEX(w, (tid_x / pass_1_blocksize));
                zc_xor_zt ^= block_intermediate_prefix_z[bid];
                xc_xor_xt ^= block_intermediate_prefix_x[bid];

                // Compute the CX expression for Z.
                word_std_t c_stab_word = zs[c_stab];
                word_std_t t_destab_word = zs[t_destab];
                xc_and_zt = (c_stab_word & t_destab_word);
                not_zc_xor_xt = ~(zc_xor_zt ^ zs[t_stab]);
                local_destab_sign ^= xc_and_zt & not_zc_xor_xt;
                
                // Update Z tableau.
                zs[t_stab] ^= c_stab_word;
                zc_destab ^= t_destab_word; // requires collapse.

                // Compute the CX expression for X.
                c_stab_word = xs[c_stab];
                t_destab_word = xs[t_destab];
                xc_and_zt = (c_stab_word & t_destab_word);
                not_zc_xor_xt = ~(xc_xor_xt ^ xs[t_stab]);
                local_stab_sign ^= xc_and_zt & not_zc_xor_xt;

                // Update X tableau.
                xs[t_stab] ^= c_stab_word;
                xc_destab ^= t_destab_word; // requires collapse.
            }

            typedef hipcub::BlockReduce<word_std_t, BLOCKX> BlockReduce;

            __shared__ typename BlockReduce::TempStorage temp_storage_zc[BLOCKY];
            __shared__ typename BlockReduce::TempStorage temp_storage_xc[BLOCKY];
            __shared__ typename BlockReduce::TempStorage temp_storage_destab_sign[BLOCKY];
            __shared__ typename BlockReduce::TempStorage temp_storage_stab_sign[BLOCKY];

            word_std_t block_zc_destab = BlockReduce(temp_storage_zc[threadIdx.y]).Reduce(zc_destab, XOROP());
            word_std_t block_xc_destab = BlockReduce(temp_storage_xc[threadIdx.y]).Reduce(xc_destab, XOROP());
            word_std_t block_local_destab_sign = BlockReduce(temp_storage_destab_sign[threadIdx.y]).Reduce(local_destab_sign, XOROP());
            word_std_t block_local_stab_sign = BlockReduce(temp_storage_stab_sign[threadIdx.y]).Reduce(local_stab_sign, XOROP());

            if (!threadIdx.x) {
                const pivot_t pivot = pivots[0];
                const size_t c_destab = TABLEAU_INDEX(w, pivot);
                if (block_zc_destab)
                    atomicXOR(zs + c_destab, block_zc_destab);
                if (block_xc_destab)
                    atomicXOR(xs + c_destab, block_xc_destab);
                if (block_local_destab_sign)
                    atomicXOR(inv_ss->data(w), block_local_destab_sign);
                if (block_local_stab_sign)
                    atomicXOR(inv_ss->data(w + num_words_minor), block_local_stab_sign);
            }
        }
    }

    #define CALL_INJECTCX_PASS_2_FOR_BLOCK(X, Y) \
        scan_targets_pass_2 <X, Y> \
        <<<currentgrid, currentblock, 0, stream>>> ( \
                XZ_TABLE(input), \
                input.signs(), \
                XZ_TABLE(targets), \
                block_intermediate_prefix_z, \
                block_intermediate_prefix_x, \
                pivots, \
                active_targets, \
                num_words_major, \
                num_words_minor, \
                num_qubits_padded, \
                max_blocks, \
                pass_1_blocksize\
            )

	void call_injectcx_pass_1_kernel(
                Tableau& 			targets, 
                Tableau& 			input,
                word_std_t *        block_intermediate_prefix_z,
                word_std_t *        block_intermediate_prefix_x,
        const   pivot_t*            pivots,
        const   size_t&             active_targets,
        const   size_t&             num_words_major,
        const   size_t&             num_words_minor,
        const   size_t&             num_qubits_padded,
        const   size_t&             max_blocks,
        const   dim3&               currentblock,
        const   dim3&               currentgrid,
        const   hipStream_t&       stream) {
        
        GENERATE_SWITCH_FOR_CALL(CALL_INJECTCX_PASS_1_FOR_BLOCK)
    }

	void call_injectcx_pass_2_kernel(
                Tableau& 			targets, 
                Tableau& 			input,
                ConstWordsPointer   block_intermediate_prefix_z,
                ConstWordsPointer   block_intermediate_prefix_x,
        const   pivot_t*            pivots,
        const   size_t&             active_targets,
        const   size_t&             num_words_major,
        const   size_t&             num_words_minor,
        const   size_t&             num_qubits_padded,
        const   size_t&             max_blocks,
        const   size_t&             pass_1_blocksize,
        const   dim3&               currentblock,
        const   dim3&               currentgrid,
        const   hipStream_t&       stream) {
        
        GENERATE_SWITCH_FOR_CALL(CALL_INJECTCX_PASS_2_FOR_BLOCK)
    }

    // We need to compute prefix-xor of t-th destabilizer in X,Z for t = c+1, c+2, ... c+n-1
    // so that later we can xor every prefix-xor with controlled destabilizer.
    void Prefix::scan_large(Tableau& input, const pivot_t* pivots, const size_t& active_targets, const hipStream_t& stream) {
        if (active_targets <= 1024) {
            LOGERROR("active targets %d are too low for large scanning", active_targets);
        }
        assert(nextPow2(MIN_BLOCK_INTERMEDIATE_SIZE) == MIN_BLOCK_INTERMEDIATE_SIZE);
        const size_t num_qubits_padded = input.num_qubits_padded();

        // Do the first phase of prefix.
        dim3 currentblock, currentgrid;
        if (bestblockinjectprepare.x == 1)
            LOGERROR("x-block size in inject-cx is 1");
        TRIM_Y_BLOCK_IN_DEBUG_MODE(bestblockinjectprepare, bestgridinjectprepare, num_words_minor);
        currentblock = bestblockinjectprepare, currentgrid = bestgridinjectprepare;
        FORCE_TRIM_GRID_IN_XY(active_targets, num_words_minor);
        const size_t pass_1_blocksize = currentblock.x;
        const size_t pass_1_gridsize = ROUNDUP(active_targets, pass_1_blocksize);
        if (pass_1_gridsize > max_intermediate_blocks)
            LOGERROR("too many blocks for intermediate arrays");
        LOGN2(2, " Running pass-1 kernel for %d targets with block(x:%u, y:%u) and grid(x:%u, y:%u).. ",
            active_targets, currentblock.x, currentblock.y, currentgrid.x, currentgrid.y);
        if (options.sync) cutimer.start(stream);
        call_injectcx_pass_1_kernel(
            targets, 
            input, 
            zblocks(), 
            xblocks(),
            pivots,
            active_targets, 
            num_words_major, 
            num_words_minor,
            num_qubits_padded,
            max_intermediate_blocks,
            currentblock,
            currentgrid,
            stream
        );
        if (options.sync) {
            LASTERR("failed to scan targets in pass 1");
            cutimer.stop(stream);
            LOGENDING(2, 4, "(time %.3f ms)", cutimer.time());
        } else LOGDONE(2, 4);

        // Verify pass-1 prefix.
        if (options.check_measurement) {
            checker.check_prefix_pass_1(
                targets,
                pivots,
                zblocks(), 
                xblocks(),
                active_targets,
                max_intermediate_blocks,
                pass_1_blocksize,
                pass_1_gridsize);
        }

        // Intermeditae scan of blocks resulted in pass 1.
        scan_blocks(nextPow2(pass_1_gridsize), pass_1_blocksize, stream);

        // Verify intermediate-pass prefix.
        if (options.check_measurement) {
            checker.check_prefix_intermediate_pass(
                zblocks(), 
                xblocks(),
                max_intermediate_blocks,
                pass_1_gridsize);
        }

        // Second phase of injecting CX.
        TRIM_BLOCK_IN_DEBUG_MODE(bestblockinjectfinal, bestgridinjectfinal, active_targets, num_words_minor);
        currentblock = bestblockinjectfinal, currentgrid = bestgridinjectfinal;
        if (currentblock.x > active_targets) {
            currentblock.x = active_targets == 1 ? 2 : MIN(currentblock.x, nextPow2(active_targets));
        }
        FORCE_TRIM_GRID_IN_XY(active_targets, num_words_minor);
        LOGN2(2, " Running pass-2 kernel for %d targets with block(x:%u, y:%u) and grid(x:%u, y:%u).. ", \
            active_targets, currentblock.x, currentblock.y, currentgrid.x, currentgrid.y); \
        if (options.sync) cutimer.start(stream);
        call_injectcx_pass_2_kernel(
            targets, 
            input,
            zblocks(), 
            xblocks(), 
            pivots, 
            active_targets, 
            num_words_major, 
            num_words_minor, 
            num_qubits_padded,
            max_intermediate_blocks,
            pass_1_blocksize,
            currentblock,
            currentgrid,
            stream
        );
        if (options.sync) {
            LASTERR("failed to scan targets in pass 2");
            cutimer.stop(stream);
            LOGENDING(2, 4, "(time %.3f ms)", cutimer.time());
        } else LOGDONE(2, 4);

        // Verify pass-2 prefix.
        if (options.check_measurement) {
            checker.check_prefix_pass_2(
                targets, 
                input,
                active_targets, 
                max_intermediate_blocks,
                pass_1_blocksize);
        }
    }

    void Prefix::tune_inject_cx(Tableau& input, const pivot_t* pivots, const size_t& max_active_targets) {
        assert(nextPow2(MIN_BLOCK_INTERMEDIATE_SIZE) == MIN_BLOCK_INTERMEDIATE_SIZE);
        const size_t num_qubits_padded = input.num_qubits_padded();

        // Do the first phase of prefix.
        if (options.tune_injectprepare) {
            SYNCALL;
            tune_inject_pass_1(
                bestblockinjectprepare, bestgridinjectprepare,
                2 * sizeof(word_std_t), // used to skip very large blocks.
                max_active_targets,
                num_words_minor,
                targets, 
                input, 
                zblocks(), 
                xblocks(),
                pivots,
                max_active_targets, 
                num_words_major, 
                num_words_minor,
                num_qubits_padded,
                max_intermediate_blocks
            );
            SYNCALL;
        }

        const size_t pass_1_blocksize = bestblockinjectprepare.x;
        const size_t pass_1_gridsize = ROUNDUP(max_active_targets, pass_1_blocksize);
        tune_scan_blocks(nextPow2(pass_1_gridsize), pass_1_blocksize);

        if (options.tune_injectfinal) {
            SYNCALL;
            tune_inject_pass_2(
                bestblockinjectfinal, bestgridinjectfinal,
                4 * sizeof(word_std_t),
                max_active_targets,
                num_words_minor,
                targets, 
                input,
                zblocks(), 
                xblocks(), 
                pivots, 
                max_active_targets, 
                num_words_major, 
                num_words_minor, 
                num_qubits_padded,
                max_intermediate_blocks,
                pass_1_blocksize
            );
            SYNCALL;
        }
    }

}