#include "hip/hip_runtime.h"

#include "prefix.cuh"
#include "collapse.cuh"
#include "access.cuh"

namespace QuaSARQ {

    __global__ 
    void scan_targets_pass_1(
        Table *             prefix_xs, 
        Table *             prefix_zs, 
        Table *             inv_xs, 
        Table *             inv_zs,
        word_std_t *        block_intermediate_prefix_z,
        word_std_t *        block_intermediate_prefix_x,
        const Commutation * commutations,
        const uint32        c,
        const size_t        total_targets,
        const size_t        num_words_major,
        const size_t        num_words_minor,
        const size_t        num_qubits_padded,
        const size_t        max_blocks)
    {
        grid_t padded_block_size = blockDim.x + CONFLICT_FREE_OFFSET(blockDim.x);
        grid_t slice = 2 * padded_block_size;
        word_std_t *shared = SharedMemory<word_std_t>();
        word_std_t *t_prefix_z = shared + threadIdx.y * slice;
        word_std_t *t_prefix_x = t_prefix_z + padded_block_size;
        grid_t prefix_tid = threadIdx.x + CONFLICT_FREE_OFFSET(threadIdx.x);
        word_t *xs = inv_xs->data();
        word_t *zs = inv_zs->data();

        for_parallel_y(w, num_words_minor) {

            const size_t c_destab = TABLEAU_INDEX(w, c);
            assert(c_destab < inv_zs->size());
            assert(c_destab < inv_xs->size());

            for_parallel_x(tid_x, total_targets) {

                const size_t t = tid_x + c + 1;
                word_std_t t_delta_z = 0;
                word_std_t t_delta_x = 0;

                if (commutations[t].anti_commuting) {
                    const size_t t_destab = TABLEAU_INDEX(w, t);
                    assert(t_destab < inv_zs->size());
                    assert(t_destab < inv_xs->size());
                    t_delta_z = zs[t_destab];
                    t_delta_x = xs[t_destab];
                }
            
                t_prefix_z[prefix_tid] = t_delta_z;
                t_prefix_x[prefix_tid] = t_delta_x;

                __syncthreads();

                word_std_t blockSum_z = scan_block_exclusive(t_prefix_z, blockDim.x);
                word_std_t blockSum_x = scan_block_exclusive(t_prefix_x, blockDim.x);

                const size_t word_idx = w * total_targets + tid_x;
                assert(word_idx < prefix_zs->size());
                assert(word_idx < prefix_xs->size());
                // Compute local zc = zc ^ zt, where zt is the zt'prefix.
                (*prefix_zs)[word_idx] = word_std_t(zs[c_destab]) ^ t_prefix_z[prefix_tid];
                // Compute local xc = xc ^ xt, where xt is the xt'prefix.
                (*prefix_xs)[word_idx] = word_std_t(xs[c_destab]) ^ t_prefix_x[prefix_tid];


                if (threadIdx.x == blockDim.x - 1) {
                    assert((blockIdx.x * num_words_minor + w) < gridDim.x * num_words_minor);
                    const size_t bid = w * max_blocks + (tid_x / blockDim.x);
                    block_intermediate_prefix_z[bid] = blockSum_z;
                    block_intermediate_prefix_x[bid] = blockSum_x;
                    // printf("w(%lld), t(%lld):  block intermediate prefix-xor (tz) = " B2B_STR "\n", w, global_tid, RB2B(block_intermediate_prefix_z[blockIdx.x * num_words_minor + w]));
                }

            }
        }
    }

    __global__ 
    void scan_targets_pass_2(
                Table *         prefix_xs, 
                Table *         prefix_zs, 
                Table *         inv_xs, 
                Table *         inv_zs,
        const   word_std_t *    block_intermediate_prefix_z,
        const   word_std_t *    block_intermediate_prefix_x,
        const   Commutation *   commutations,
        const   uint32          c,
        const   size_t          total_targets,
        const   size_t          num_words_major,
        const   size_t          num_words_minor,
        const   size_t          num_qubits_padded,
        const   size_t          max_blocks,
        const   size_t          phase1_block_size)
    { 
        word_std_t *shared = SharedMemory<word_std_t>();
        word_std_t *shared_z = shared;
        word_std_t *shared_x = shared_z + blockDim.x;
        grid_t collapse_tid = threadIdx.y * 2 * blockDim.x + threadIdx.x;
        word_std_t *xs = inv_xs->words();
        word_std_t *zs = inv_zs->words();

        for_parallel_y(w, num_words_minor) {

            const size_t c_destab = TABLEAU_INDEX(w, c);
            const size_t c_stab = c_destab + TABLEAU_STAB_OFFSET;

            // For parallel collapsing.
            word_std_t zc_destab = 0;
            word_std_t xc_destab = 0;
            word_std_t xc_and_zt = 0;
            word_std_t not_zc_xor_xt = 0;
            word_std_t local_destab_sign = 0;
            word_std_t local_stab_sign = 0;

            for_parallel_x(tid_x, total_targets) {

                size_t t = tid_x + c + 1;

                if (commutations[t].anti_commuting) {

                    const size_t t_destab = TABLEAU_INDEX(w, t);
                    const size_t t_stab = t_destab + TABLEAU_STAB_OFFSET;

                    assert(c_destab < inv_zs->size());
                    assert(t_destab < inv_zs->size());

                    const size_t word_idx = w * total_targets + tid_x;
                    word_std_t zc_xor_zt = (*prefix_zs)[word_idx];
                    word_std_t xc_xor_xt = (*prefix_xs)[word_idx];

                    // Compute final prefixes and hence final {x,z}'c = {x,z}'c ^ {x,z}'t expressions.
                    const size_t bid = w * max_blocks + (tid_x / phase1_block_size);
                    zc_xor_zt ^= block_intermediate_prefix_z[bid];
                    xc_xor_xt ^= block_intermediate_prefix_x[bid];

                    // Compute the CX expression for Z.
                    word_std_t c_stab_word = zs[c_stab];
                    word_std_t t_destab_word = zs[t_destab];
                    xc_and_zt = (c_stab_word & t_destab_word);
                    not_zc_xor_xt = ~(zc_xor_zt ^ zs[t_stab]);

                    (*prefix_zs)[word_idx] = xc_and_zt & not_zc_xor_xt;

                    // Update Z tableau.
                    zs[t_stab] ^= c_stab_word;
                    zc_destab ^= t_destab_word; // requires collapse.

                    // Compute the CX expression for X.
                    c_stab_word = xs[c_stab];
                    t_destab_word = xs[t_destab];
                    xc_and_zt = (c_stab_word & t_destab_word);
                    not_zc_xor_xt = ~(xc_xor_xt ^ xs[t_stab]);

                    (*prefix_xs)[word_idx] = xc_and_zt & not_zc_xor_xt;

                    // Update X tableau.
                    xs[t_stab] ^= c_stab_word;
                    xc_destab ^= t_destab_word; // requires collapse.
                }
            }

            // Update Z, X in shared memory.
            collapse_load_shared_dual(shared_z, zc_destab, shared_x, xc_destab, collapse_tid, total_targets);
            collapse_shared_dual(shared_z, zc_destab, shared_x, xc_destab, collapse_tid);
            collapse_warp_dual(shared_z, zc_destab, shared_x, xc_destab, collapse_tid);

            if (!threadIdx.x) {
                if (zc_destab)
                    atomicXOR(zs + c_destab, zc_destab);
                if (xc_destab)
                    atomicXOR(xs + c_destab, xc_destab);
            }
        }
    }

    __global__ 
    void collapse_scanned_targets(
                Table *         prefix_xs, 
                Table *         prefix_zs, 
                Table *         inv_xs, 
                Table *         inv_zs, 
                Signs *         inv_ss,
        const   Commutation *   commutations,
        const   uint32          c,
        const   size_t          total_targets,
        const   size_t          num_words_major, 
        const   size_t          num_words_minor,
        const   size_t          num_qubits_padded)
    {
        word_t *xs = inv_xs->data();
        word_t *zs = inv_zs->data();
        sign_t *ss = inv_ss->data();
        word_std_t *signs_destab = SharedMemory<word_std_t>();
        word_std_t *signs_stab = signs_destab + blockDim.x;

        grid_t collapse_tid = threadIdx.y * 2 * blockDim.x + threadIdx.x;

        for_parallel_y(w, num_words_minor) { 

            word_std_t local_destab_sign = 0;
            word_std_t local_stab_sign = 0;

            for_parallel_x(tid_x, total_targets) {
                size_t t = tid_x + c + 1;
                if (commutations[t].anti_commuting) {
                    const size_t word_idx = w * total_targets + tid_x;
                    local_destab_sign ^= (word_std_t)(*prefix_zs)[word_idx];
                    local_stab_sign ^= (word_std_t)(*prefix_xs)[word_idx];
                }
            }

            collapse_load_shared_dual(signs_destab, local_destab_sign, signs_stab, local_stab_sign, collapse_tid, total_targets);
            collapse_shared_dual(signs_destab, local_destab_sign, signs_stab, local_stab_sign, collapse_tid);
            collapse_warp_dual(signs_destab, local_destab_sign, signs_stab, local_stab_sign, collapse_tid);

            if (!threadIdx.x && local_destab_sign)
                atomicXOR(inv_ss->data(w), local_destab_sign);
            if (!threadIdx.x && local_stab_sign)
                atomicXOR(inv_ss->data(w + num_words_minor), local_stab_sign);
        }
    }

    // We need to compute prefix-xor of t-th destabilizer in X,Z for t = c+1, c+2, ... c+n-1
    // so that later we can xor every prefix-xor with controlled destabilizer.
    void Prefix::inject_CX(Tableau<DeviceAllocator>& input, const Commutation* commutations, const uint32& pivot, const qubit_t& qubit, const hipStream_t& stream) {
        assert(num_qubits > pivot);
        assert(nextPow2(MIN_BLOCK_INTERMEDIATE_SIZE) == MIN_BLOCK_INTERMEDIATE_SIZE);
        
        const size_t num_qubits_padded = input.num_qubits_padded();

        // Calculate number of target generators.
        const size_t total_targets = num_qubits - pivot - 1;
        if (!total_targets) return;
        // Do the first phase of prefix.
        dim3 currentblock, currentgrid;
        if (options.tune_injectprepare) {
            SYNCALL;
            tune_inject_pass_1(
                scan_targets_pass_1, 
                bestblockinjectprepare, bestgridinjectprepare,
                2 * sizeof(word_std_t),
                total_targets,
                num_words_minor,
                XZ_TABLE(targets), 
                XZ_TABLE(input), 
                zblocks(), 
                xblocks(),
                commutations, 
                pivot,
                total_targets, 
                num_words_major, 
                num_words_minor,
                num_qubits_padded,
                max_intermediate_blocks
            );
            SYNCALL;
        }
        SYNCALL;
        TRIM_BLOCK_IN_DEBUG_MODE(bestblockinjectprepare, bestgridinjectprepare, total_targets, num_words_minor);
        currentblock = bestblockinjectprepare, currentgrid = bestgridinjectprepare;
        TRIM_GRID_IN_XY(total_targets, num_words_minor);
        const size_t pass_1_blocksize = currentblock.x;
        const size_t pass_1_gridsize = ROUNDUP(total_targets, pass_1_blocksize);
        if (pass_1_gridsize > max_intermediate_blocks)
            LOGERROR("too many blocks for intermediate arrays.");
        OPTIMIZESHARED(smem_size, currentblock.y * (currentblock.x + CONFLICT_FREE_OFFSET(currentblock.x)), 2 * sizeof(word_std_t));
        scan_targets_pass_1 <<<currentgrid, currentblock, smem_size, stream>>> (
                    XZ_TABLE(targets), 
                    XZ_TABLE(input), 
                    zblocks(), 
                    xblocks(),
                    commutations, 
                    pivot,
                    total_targets, 
                    num_words_major, 
                    num_words_minor,
                    num_qubits_padded,
                    max_intermediate_blocks
                );
        if (options.sync) {
            LASTERR("failed to scan targets in pass 1");
            SYNC(stream);
        }

        // Intermeditae scan of blocks resulted in pass 1.
        scan_blocks(nextPow2(pass_1_gridsize), stream);

        // Second phase of injecting CX.
        if (options.tune_injectfinal) {
            SYNCALL;
            tune_inject_pass_2(
                scan_targets_pass_2, 
                bestblockinjectfinal, bestgridinjectfinal,
                2 * sizeof(word_std_t),
                total_targets,
                num_words_minor,
                XZ_TABLE(targets), 
                XZ_TABLE(input),
                zblocks(), 
                xblocks(), 
                commutations, 
                pivot, 
                total_targets, 
                num_words_major, 
                num_words_minor, 
                num_qubits_padded,
                max_intermediate_blocks,
                pass_1_blocksize
            );
            SYNCALL;
        }
        TRIM_BLOCK_IN_DEBUG_MODE(bestblockinjectfinal, bestgridinjectfinal, total_targets, num_words_minor);
        currentblock = bestblockinjectfinal, currentgrid = bestgridinjectfinal;
        TRIM_GRID_IN_XY(total_targets, num_words_minor);
        OPTIMIZESHARED(finalize_prefix_smem_size, currentblock.y * currentblock.x, 2 * sizeof(word_std_t));
        scan_targets_pass_2 <<<currentgrid, currentblock, finalize_prefix_smem_size, stream>>> (
                        XZ_TABLE(targets), 
                        XZ_TABLE(input),
                        zblocks(), 
                        xblocks(), 
                        commutations, 
                        pivot, 
                        total_targets, 
                        num_words_major, 
                        num_words_minor, 
                        num_qubits_padded,
                        max_intermediate_blocks,
                        pass_1_blocksize
                    );
        if (options.sync) {
            LASTERR("failed to scan targets in pass 2");
            SYNC(stream);
        }

        // Final phase to compute the signs of the scanned targets.
        if (options.tune_collapsetargets) {
            SYNCALL;
            tune_collapse_targets(
                collapse_scanned_targets, 
                bestblockcollapsetargets, bestgridcollapsetargets,
                2 * sizeof(word_std_t),
                total_targets,
                num_words_minor,
                XZ_TABLE(targets), 
                XZ_TABLE(input), 
                input.signs(), 
                commutations, 
                pivot, 
                total_targets, 
                num_words_major, 
                num_words_minor,
                num_qubits_padded
            );
            SYNCALL;
        }
        TRIM_BLOCK_IN_DEBUG_MODE(bestblockcollapsetargets, bestgridcollapsetargets, total_targets, num_words_minor);
        currentblock = bestblockcollapsetargets, currentgrid = bestgridcollapsetargets;
        TRIM_GRID_IN_XY(total_targets, num_words_minor);
        OPTIMIZESHARED(reduce_smem_size, currentblock.y * currentblock.x, 2 * sizeof(word_std_t));
        collapse_scanned_targets <<<currentgrid, currentblock, reduce_smem_size, stream>>> (
                        XZ_TABLE(targets), 
                        XZ_TABLE(input), 
                        input.signs(), 
                        commutations, 
                        pivot, 
                        total_targets, 
                        num_words_major, 
                        num_words_minor,
                        num_qubits_padded
                    );
        if (options.sync) {
            LASTERR("failed to collapse scanned targets");
            SYNC(stream);
        }
    }

}