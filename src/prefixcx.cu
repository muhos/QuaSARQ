#include "hip/hip_runtime.h"

#include "prefix.cuh"
#include "collapse.cuh"
#include "access.cuh"
#include "vector.hpp"
#include "print.cuh"
#include "prefixcxcub.cuh"
#include "prefixdim.cuh"

namespace QuaSARQ {

    __global__ 
    void scan_targets_pass_1(
                Table *             prefix_xs, 
                Table *             prefix_zs, 
                Table *             inv_xs, 
                Table *             inv_zs,
                word_std_t *        block_intermediate_prefix_z,
                word_std_t *        block_intermediate_prefix_x,
        const   Commutation *       commutations,
        const   uint32              pivot,
        const   size_t              total_targets,
        const   size_t              num_words_major,
        const   size_t              num_words_minor,
        const   size_t              num_qubits_padded,
        const   size_t              max_blocks)
    {
        grid_t padded_block_size = blockDim.x + CONFLICT_FREE_OFFSET(blockDim.x);
        grid_t slice = 2 * padded_block_size;
        word_std_t *shared = SharedMemory<word_std_t>();
        word_std_t *t_prefix_z = shared + threadIdx.y * slice;
        word_std_t *t_prefix_x = t_prefix_z + padded_block_size;
        grid_t prefix_tid = threadIdx.x + CONFLICT_FREE_OFFSET(threadIdx.x);

        for_parallel_y_tiled(by, num_words_minor) {

            const grid_t w = threadIdx.y + by * blockDim.y;

            for_parallel_x_tiled(bx, total_targets) {

                const grid_t tid_x = threadIdx.x + bx * blockDim.x;
                
                word_std_t z = 0;
                word_std_t x = 0;

                if (w < num_words_minor && tid_x < total_targets) {
                    const size_t t = tid_x + pivot + 1;
                    if (commutations[t].anti_commuting) {
                        const size_t t_destab = TABLEAU_INDEX(w, t);
                        z = (*inv_zs)[t_destab];
                        x = (*inv_xs)[t_destab];
                    }
                }

                t_prefix_z[prefix_tid] = z;
                t_prefix_x[prefix_tid] = x;

                __syncthreads();

                word_std_t blockSum_z = scan_block_exclusive(t_prefix_z, blockDim.x);
                word_std_t blockSum_x = scan_block_exclusive(t_prefix_x, blockDim.x);

                if (w < num_words_minor && tid_x < total_targets) {
                    const size_t word_idx = PREFIX_TABLEAU_INDEX(w, tid_x);
                    assert(word_idx < prefix_zs->size());
                    assert(word_idx < prefix_xs->size());
                    size_t c_destab = TABLEAU_INDEX(w, pivot);
                    (*prefix_zs)[word_idx] = word_std_t((*inv_zs)[c_destab]) ^ t_prefix_z[prefix_tid];
                    (*prefix_xs)[word_idx] = word_std_t((*inv_xs)[c_destab]) ^ t_prefix_x[prefix_tid];
                }

                if (w < num_words_minor && threadIdx.x == blockDim.x - 1) {
                    assert((blockIdx.x * num_words_minor + w) < gridDim.x * num_words_minor);
                    const size_t bid = w * max_blocks + bx;
                    block_intermediate_prefix_z[bid] = blockSum_z;
                    block_intermediate_prefix_x[bid] = blockSum_x;
                }
            }
        }
    }

    __global__ 
    void scan_targets_pass_2(
                Table *         prefix_xs, 
                Table *         prefix_zs, 
                Table *         inv_xs, 
                Table *         inv_zs,
                Signs *         inv_ss,
        const   word_std_t *    block_intermediate_prefix_z,
        const   word_std_t *    block_intermediate_prefix_x,
        const   Commutation *   commutations,
        const   uint32          pivot,
        const   size_t          total_targets,
        const   size_t          num_words_major,
        const   size_t          num_words_minor,
        const   size_t          num_qubits_padded,
        const   size_t          max_blocks,
        const   size_t          pass_1_blocksize)
    { 
        word_std_t *shared = SharedMemory<word_std_t>();
        word_std_t *shared_z = shared;
        word_std_t *shared_x = shared_z + blockDim.x;
        word_std_t *signs_destab = shared_x + blockDim.x;
        word_std_t *signs_stab = signs_destab + blockDim.x;
        grid_t      collapse_tid = threadIdx.y * 4 * blockDim.x + threadIdx.x;
        word_std_t *xs = inv_xs->words();
        word_std_t *zs = inv_zs->words();

        for_parallel_y(w, num_words_minor) {

            const size_t c_destab = TABLEAU_INDEX(w, pivot);
            const size_t c_stab = c_destab + TABLEAU_STAB_OFFSET;

            word_std_t zc_destab = 0;
            word_std_t xc_destab = 0;
            word_std_t xc_and_zt = 0;
            word_std_t not_zc_xor_xt = 0;
            word_std_t local_destab_sign = 0;
            word_std_t local_stab_sign = 0;

            for_parallel_x(tid_x, total_targets) {

                size_t t = tid_x + pivot + 1;

                if (commutations[t].anti_commuting) {

                    const size_t t_destab = TABLEAU_INDEX(w, t);
                    const size_t t_stab = t_destab + TABLEAU_STAB_OFFSET;

                    assert(c_destab < inv_zs->size());
                    assert(t_destab < inv_zs->size());

                    const size_t word_idx = PREFIX_TABLEAU_INDEX(w, tid_x);
                    word_std_t zc_xor_zt = (*prefix_zs)[word_idx];
                    word_std_t xc_xor_xt = (*prefix_xs)[word_idx];

                    // Compute final prefixes and hence final {x,z}'c = {x,z}'c ^ {x,z}'t expressions.
                    const size_t bid = w * max_blocks + (tid_x / pass_1_blocksize);
                    zc_xor_zt ^= block_intermediate_prefix_z[bid];
                    xc_xor_xt ^= block_intermediate_prefix_x[bid];

                    // Compute the CX expression for Z.
                    word_std_t c_stab_word = zs[c_stab];
                    word_std_t t_destab_word = zs[t_destab];
                    xc_and_zt = (c_stab_word & t_destab_word);
                    not_zc_xor_xt = ~(zc_xor_zt ^ zs[t_stab]);
                    local_destab_sign ^= xc_and_zt & not_zc_xor_xt;
                    
                    // Update Z tableau.
                    zs[t_stab] ^= c_stab_word;
                    zc_destab ^= t_destab_word; // requires collapse.

                    // Compute the CX expression for X.
                    c_stab_word = xs[c_stab];
                    t_destab_word = xs[t_destab];
                    xc_and_zt = (c_stab_word & t_destab_word);
                    not_zc_xor_xt = ~(xc_xor_xt ^ xs[t_stab]);
                    local_stab_sign ^= xc_and_zt & not_zc_xor_xt;

                    // Update X tableau.
                    xs[t_stab] ^= c_stab_word;
                    xc_destab ^= t_destab_word; // requires collapse.
                }
            }

            collapse_load_shared_dual(shared_z, zc_destab, shared_x, xc_destab, collapse_tid, total_targets);
            collapse_shared_dual(shared_z, zc_destab, shared_x, xc_destab, collapse_tid);
            collapse_warp_dual(shared_z, zc_destab, shared_x, xc_destab, collapse_tid);
            collapse_load_shared_dual(signs_destab, local_destab_sign, signs_stab, local_stab_sign, collapse_tid, total_targets);
            collapse_shared_dual(signs_destab, local_destab_sign, signs_stab, local_stab_sign, collapse_tid);
            collapse_warp_dual(signs_destab, local_destab_sign, signs_stab, local_stab_sign, collapse_tid);

            if (!threadIdx.x) {
                if (zc_destab)
                    atomicXOR(zs + c_destab, zc_destab);
                if (xc_destab)
                    atomicXOR(xs + c_destab, xc_destab);
                if (local_destab_sign)
                    atomicXOR(inv_ss->data(w), local_destab_sign);
                if (local_stab_sign)
                    atomicXOR(inv_ss->data(w + num_words_minor), local_stab_sign);
            }
        }
    }

    void call_pass_1_kernel(
                Tableau<DeviceAllocator>& targets, 
                Tableau<DeviceAllocator>& input,
                word_std_t *        block_intermediate_prefix_z,
                word_std_t *        block_intermediate_prefix_x,
        const   Commutation *       commutations,
        const   uint32              pivot,
        const   size_t              total_targets,
        const   size_t              num_words_major,
        const   size_t              num_words_minor,
        const   size_t              num_qubits_padded,
        const   size_t              max_blocks,
        const   dim3&               currentblock,
        const   dim3&               currentgrid,
        const   hipStream_t&       stream) {
        
        switch (currentblock.y) {
            POW2_Y_DIM_1(CALL_PASS_1_FOR_BLOCK);
            POW2_Y_DIM_2(CALL_PASS_1_FOR_BLOCK);
            POW2_Y_DIM_4(CALL_PASS_1_FOR_BLOCK);
            POW2_Y_DIM_8(CALL_PASS_1_FOR_BLOCK);
            POW2_Y_DIM_16(CALL_PASS_1_FOR_BLOCK);
            POW2_Y_DIM_32(CALL_PASS_1_FOR_BLOCK);
            default:
            LOGERROR("unknown block size in y-dimension");
        }
    }

    // We need to compute prefix-xor of t-th destabilizer in X,Z for t = c+1, c+2, ... c+n-1
    // so that later we can xor every prefix-xor with controlled destabilizer.
    void Prefix::inject_CX(Tableau<DeviceAllocator>& input, const Commutation* commutations, const uint32& pivot, const qubit_t& qubit, const hipStream_t& stream) {
        assert(num_qubits > pivot);
        assert(nextPow2(MIN_BLOCK_INTERMEDIATE_SIZE) == MIN_BLOCK_INTERMEDIATE_SIZE);
        
        const size_t num_qubits_padded = input.num_qubits_padded();

        // Calculate number of target generators.
        const size_t total_targets = num_qubits - pivot - 1;
        if (!total_targets) return;

        // Do the first phase of prefix.
        dim3 currentblock, currentgrid;
        if (options.tune_injectprepare) {
            SYNCALL;
            // tune_inject_pass_1(
            //     scan_targets_pass_1, 
            //     bestblockinjectprepare, bestgridinjectprepare,
            //     2 * sizeof(word_std_t),
            //     total_targets,
            //     num_words_minor,
            //     XZ_TABLE(targets), 
            //     XZ_TABLE(input), 
            //     zblocks(), 
            //     xblocks(),
            //     commutations, 
            //     pivot,
            //     total_targets, 
            //     num_words_major, 
            //     num_words_minor,
            //     num_qubits_padded,
            //     max_intermediate_blocks
            // );
            tune_inject_pass_1(
                bestblockinjectprepare, bestgridinjectprepare,
                2 * sizeof(word_std_t),
                total_targets,
                num_words_minor,
                targets, 
                input, 
                zblocks(), 
                xblocks(),
                commutations, 
                pivot,
                total_targets, 
                num_words_major, 
                num_words_minor,
                num_qubits_padded,
                max_intermediate_blocks
            );
            SYNCALL;
        }
        TRIM_Y_BLOCK_IN_DEBUG_MODE(bestblockinjectprepare, bestgridinjectprepare, num_words_minor);
        currentblock = bestblockinjectprepare, currentgrid = bestgridinjectprepare;
        TRIM_GRID_IN_XY(total_targets, num_words_minor);
        const size_t pass_1_blocksize = currentblock.x;
        const size_t pass_1_gridsize = ROUNDUP(total_targets, pass_1_blocksize);
        if (pass_1_gridsize > max_intermediate_blocks)
            LOGERROR("too many blocks for intermediate arrays.");
        call_pass_1_kernel(
            targets, 
            input, 
            zblocks(), 
            xblocks(),
            commutations, 
            pivot,
            total_targets, 
            num_words_major, 
            num_words_minor,
            num_qubits_padded,
            max_intermediate_blocks,
            currentblock,
            currentgrid,
            stream
        );
        //OPTIMIZESHARED(smem_size, currentblock.y * (currentblock.x + CONFLICT_FREE_OFFSET(currentblock.x)), 2 * sizeof(word_std_t));
        //scan_targets_pass_1 <<<currentgrid, currentblock, smem_size, stream>>> (
                //     XZ_TABLE(targets), 
                //     XZ_TABLE(input), 
                //     zblocks(), 
                //     xblocks(),
                //     commutations, 
                //     pivot,
                //     total_targets, 
                //     num_words_major, 
                //     num_words_minor,
                //     num_qubits_padded,
                //     max_intermediate_blocks
                // );
        if (options.sync) {
            LASTERR("failed to scan targets in pass 1");
            SYNC(stream);
        }

        // Verify pass-1 prefix.
        assert(checker.check_prefix_pass_1(
            targets,
            input,
            commutations,
            zblocks(), 
            xblocks(),
            qubit,
            pivot,
            total_targets,
            num_words_major,
            num_words_minor,
            num_qubits_padded,
            max_intermediate_blocks,
            pass_1_blocksize,
            pass_1_gridsize
        ));

        // Intermeditae scan of blocks resulted in pass 1.
        scan_blocks(nextPow2(pass_1_gridsize), pass_1_blocksize, stream);

        // Verify intermediate-pass prefix.
        assert(checker.check_prefix_intermediate_pass(
            zblocks(), 
            xblocks(),
            qubit,
            pivot,
            num_words_minor,
            max_intermediate_blocks,
            pass_1_gridsize
        ));

        // Second phase of injecting CX.
        if (options.tune_injectfinal) {
            SYNCALL;
            tune_inject_pass_2(
                scan_targets_pass_2, 
                bestblockinjectfinal, bestgridinjectfinal,
                4 * sizeof(word_std_t),
                total_targets,
                num_words_minor,
                XZ_TABLE(targets), 
                XZ_TABLE(input),
                input.signs(),
                zblocks(), 
                xblocks(), 
                commutations, 
                pivot, 
                total_targets, 
                num_words_major, 
                num_words_minor, 
                num_qubits_padded,
                max_intermediate_blocks,
                pass_1_blocksize
            );
            SYNCALL;
        }
        TRIM_BLOCK_IN_DEBUG_MODE(bestblockinjectfinal, bestgridinjectfinal, total_targets, num_words_minor);
        currentblock = bestblockinjectfinal, currentgrid = bestgridinjectfinal;
        TRIM_GRID_IN_XY(total_targets, num_words_minor);
        OPTIMIZESHARED(finalize_prefix_smem_size, currentblock.y * currentblock.x, 4 * sizeof(word_std_t));
        LOGN2(2, " Running pass-2 kernel with block(x:%u, y:%u) and grid(x:%u, y:%u).. ", \
            currentblock.x, currentblock.y, currentgrid.x, currentgrid.y); \
        scan_targets_pass_2 
        <<<currentgrid, currentblock, finalize_prefix_smem_size, stream>>> (
            XZ_TABLE(targets), 
            XZ_TABLE(input),
            input.signs(),
            zblocks(), 
            xblocks(), 
            commutations, 
            pivot, 
            total_targets, 
            num_words_major, 
            num_words_minor, 
            num_qubits_padded,
            max_intermediate_blocks,
            pass_1_blocksize
        );
        if (options.sync) {
            LASTERR("failed to scan targets in pass 2");
            SYNC(stream);
        }
        LOGDONE(2, 4);
        // Verify pass-2 prefix.
        assert(checker.check_prefix_pass_2(
            targets, 
            input,
            qubit,
            pivot, 
            total_targets, 
            num_words_major, 
            num_words_minor, 
            num_qubits_padded,
            max_intermediate_blocks,
            pass_1_blocksize
        ));
    }

}