#include "hip/hip_runtime.h"

#include "prefix.cuh"
#include "collapse.cuh"
#include "access.cuh"
#include "vector.hpp"
#include "print.cuh"
#include "templatedim.cuh"
#include "datatypes.cuh"
#include "warp.cuh"
#include <cub/block/block_scan.cuh>
#include <cub/block/block_reduce.cuh>

namespace QuaSARQ {


    template <int BLOCKX, int BLOCKY>
    __global__ 
    void scan_targets_pass_1(
                Table *             prefix_xs, 
                Table *             prefix_zs, 
                word_std_t *        block_intermediate_prefix_z,
                word_std_t *        block_intermediate_prefix_x,
                Table*              inv_xs, 
                Table*              inv_zs,
                CPivotsPtr          pivots,
        const   size_t              active_targets,
        const   size_t              num_words_major,
        const   size_t              num_words_minor,
        const   size_t              num_qubits_padded,
        const   size_t              max_blocks) {

        assert(active_targets > 0);
        assert(BLOCKX == blockDim.x);
        assert(BLOCKY == blockDim.y);

        for_parallel_y_tiled(by, num_words_minor) {
            const grid_t w = threadIdx.y + by * BLOCKY;

            for_parallel_x_tiled(bx, active_targets) {
                const grid_t tid_x = threadIdx.x + bx * BLOCKX;
                
                word_std_t z = 0;
                word_std_t x = 0;
                word_std_t init_z = 0;
                word_std_t init_x = 0;

                if (w < num_words_minor && tid_x < active_targets) {
                    const pivot_t pivot = pivots[0];
                    assert(pivot != INVALID_PIVOT);
                    const size_t t = pivots[tid_x + 1];
                    assert(t != pivot);
                    assert(t != INVALID_PIVOT);
                    const size_t t_destab = TABLEAU_INDEX(w, t);
                    z = (*inv_zs)[t_destab];
                    x = (*inv_xs)[t_destab];
                    const size_t c_destab = TABLEAU_INDEX(w, pivot);
                    init_z = (*inv_zs)[c_destab];
                    init_x = (*inv_xs)[c_destab];
                }

                typedef hipcub::BlockScan<word_std_t, BLOCKX, hipcub::BLOCK_SCAN_RAKING> BlockScan;

                __shared__ typename BlockScan::TempStorage shared_prefix_zs[BLOCKY];
                __shared__ typename BlockScan::TempStorage shared_prefix_xs[BLOCKY];

                word_std_t blocksum_z;
                word_std_t blocksum_x;

                BlockScan(shared_prefix_zs[threadIdx.y]).ExclusiveScan(z, z, 0, XOROP(), blocksum_z);
                BlockScan(shared_prefix_xs[threadIdx.y]).ExclusiveScan(x, x, 0, XOROP(), blocksum_x);

                if (w < num_words_minor && tid_x < active_targets) {
                    const size_t word_idx = PREFIX_TABLEAU_INDEX(w, tid_x);
                    assert(word_idx < prefix_zs->size());
                    assert(word_idx < prefix_xs->size());
                    (*prefix_zs)[word_idx] = init_z ^ z;
                    (*prefix_xs)[word_idx] = init_x ^ x;
                }

                if (w < num_words_minor && threadIdx.x == BLOCKX - 1) {
                    assert((blockIdx.x * num_words_minor + w) < gridDim.x * num_words_minor);
                    const size_t bid = PREFIX_INTERMEDIATE_INDEX(w, bx);
                    block_intermediate_prefix_z[bid] = blocksum_z;
                    block_intermediate_prefix_x[bid] = blocksum_x;
                    const size_t c_destab = TABLEAU_INDEX(w, pivots[0]);
                    if (blocksum_z)
                        atomicXOR(inv_zs->words() + c_destab, blocksum_z);
                    if (blocksum_x)
                        atomicXOR(inv_xs->words() + c_destab, blocksum_x);
                }
            }
        }
    }

    #define CALL_INJECTCX_PASS_1_FOR_BLOCK(X, Y) \
        scan_targets_pass_1 <X, Y> \
        <<<currentgrid, currentblock, 0, stream>>> ( \
                XZ_TABLE(targets), \
                block_intermediate_prefix_z, \
                block_intermediate_prefix_x, \
                XZ_TABLE(input), \
                pivots, \
                active_targets, \
                num_words_major, \
                num_words_minor, \
                num_qubits_padded, \
                max_blocks \
            )

    template <int BLOCKX, int BLOCKY>
    __global__ 
    void scan_targets_pass_2(
                Table *             inv_xs, 
                Table *             inv_zs,
                Signs *             inv_ss,
                ConstTablePointer   prefix_xs, 
                ConstTablePointer   prefix_zs, 
                ConstWordsPointer   block_intermediate_prefix_z,
                ConstWordsPointer   block_intermediate_prefix_x,
                CPivotsPtr          pivots,
        const   size_t              active_targets,
        const   size_t              num_words_major,
        const   size_t              num_words_minor,
        const   size_t              num_qubits_padded,
        const   size_t              max_blocks,
        const   size_t              pass_1_blocksize)
    { 
        assert(BLOCKX == blockDim.x);
        assert(BLOCKY == blockDim.y);
        word_std_t *xs = inv_xs->words();
        word_std_t *zs = inv_zs->words();

        for_parallel_y(w, num_words_minor) {

            sign_t local_destab_sign = 0;
            sign_t local_stab_sign = 0;

            for_parallel_x(tid_x, active_targets) {
                const pivot_t pivot = pivots[0];
                assert(pivot != INVALID_PIVOT);
                const size_t t = pivots[tid_x + 1];
                assert(t != pivot);
                assert(t != INVALID_PIVOT);

                const size_t c_stab = TABLEAU_INDEX(w, pivot) + TABLEAU_STAB_OFFSET;
                const size_t t_destab = TABLEAU_INDEX(w, t);
                const size_t t_stab = t_destab + TABLEAU_STAB_OFFSET;

                const size_t word_idx = PREFIX_TABLEAU_INDEX(w, tid_x);
                word_std_t zc_xor_prefix = (*prefix_zs)[word_idx];
                word_std_t xc_xor_prefix = (*prefix_xs)[word_idx];

                // Compute final prefixes and hence final {x,z}'c = {x,z}'c ^ {x,z}'t expressions.
                const size_t bid = PREFIX_INTERMEDIATE_INDEX(w, (tid_x / pass_1_blocksize));
                zc_xor_prefix ^= block_intermediate_prefix_z[bid];
                xc_xor_prefix ^= block_intermediate_prefix_x[bid];

                compute_local_sign_per_block(local_destab_sign, zs[t_stab], zc_xor_prefix, zs[c_stab], zs[t_destab]);
                compute_local_sign_per_block(local_stab_sign, xs[t_stab], xc_xor_prefix, xs[c_stab], xs[t_destab]);
            }

            typedef hipcub::BlockReduce<sign_t, BLOCKX> BlockReduce;

            __shared__ typename BlockReduce::TempStorage shared_destab_ss[BLOCKY];
            __shared__ typename BlockReduce::TempStorage shared_stab_ss  [BLOCKY];

            sign_t block_destab_sign = BlockReduce(shared_destab_ss[threadIdx.y]).Reduce(local_destab_sign, XOROP());
            sign_t block_stab_sign   = BlockReduce(shared_stab_ss  [threadIdx.y]).Reduce(local_stab_sign,   XOROP());

            if (!threadIdx.x) {
                if (block_destab_sign)
                    atomicXOR(inv_ss->data(w), block_destab_sign);
                if (block_stab_sign)
                    atomicXOR(inv_ss->data(w + num_words_minor), block_stab_sign);
            }
        }
    }

    #define CALL_INJECTCX_PASS_2_FOR_BLOCK(X, Y) \
        scan_targets_pass_2 <X, Y> \
        <<<currentgrid, currentblock, 0, stream>>> ( \
                XZ_TABLE(input), \
                input.signs(), \
                XZ_TABLE(targets), \
                block_intermediate_prefix_z, \
                block_intermediate_prefix_x, \
                pivots, \
                active_targets, \
                num_words_major, \
                num_words_minor, \
                num_qubits_padded, \
                max_blocks, \
                pass_1_blocksize\
            )

	void call_injectcx_pass_1_kernel(
                Tableau& 			targets, 
                Tableau& 			input,
                word_std_t *        block_intermediate_prefix_z,
                word_std_t *        block_intermediate_prefix_x,
        const   pivot_t*            pivots,
        const   size_t&             active_targets,
        const   size_t&             num_words_major,
        const   size_t&             num_words_minor,
        const   size_t&             num_qubits_padded,
        const   size_t&             max_blocks,
        const   dim3&               currentblock,
        const   dim3&               currentgrid,
        const   hipStream_t&       stream) {
        
        GENERATE_SWITCH_FOR_CALL(CALL_INJECTCX_PASS_1_FOR_BLOCK)
    }

	void call_injectcx_pass_2_kernel(
                Tableau& 			targets, 
                Tableau& 			input,
                ConstWordsPointer   block_intermediate_prefix_z,
                ConstWordsPointer   block_intermediate_prefix_x,
        const   pivot_t*            pivots,
        const   size_t&             active_targets,
        const   size_t&             num_words_major,
        const   size_t&             num_words_minor,
        const   size_t&             num_qubits_padded,
        const   size_t&             max_blocks,
        const   size_t&             pass_1_blocksize,
        const   dim3&               currentblock,
        const   dim3&               currentgrid,
        const   hipStream_t&       stream) {
        
        GENERATE_SWITCH_FOR_CALL(CALL_INJECTCX_PASS_2_FOR_BLOCK)
    }

    // We need to compute prefix-xor of t-th destabilizer in X,Z for t = c+1, c+2, ... c+n-1
    // so that later we can xor every prefix-xor with controlled destabilizer.
    void Prefix::scan_large(Tableau& input, const pivot_t* pivots, const size_t& active_targets, const hipStream_t& stream) {
        assert(nextPow2(MIN_BLOCK_INTERMEDIATE_SIZE) == MIN_BLOCK_INTERMEDIATE_SIZE);
        const size_t num_qubits_padded = input.num_qubits_padded();

        // Do the first phase of prefix.
        dim3 currentblock, currentgrid;
        if (bestblockinjectprepare.x == 1)
            LOGERROR("x-block size in inject-cx is 1");
        TRIM_Y_BLOCK_IN_DEBUG_MODE(bestblockinjectprepare, bestgridinjectprepare, num_words_minor);
        currentblock = bestblockinjectprepare, currentgrid = bestgridinjectprepare;
        FORCE_TRIM_GRID_IN_XY(active_targets, num_words_minor);
        const size_t pass_1_blocksize = currentblock.x;
        const size_t pass_1_gridsize = ROUNDUP(active_targets, pass_1_blocksize);
        if (pass_1_gridsize > max_intermediate_blocks)
            LOGERROR("too many blocks for intermediate arrays");
        LOGN2(2, " Running pass-1 kernel for %d targets with block(x:%u, y:%u) and grid(x:%u, y:%u).. ",
            active_targets, currentblock.x, currentblock.y, currentgrid.x, currentgrid.y);
        if (options.sync) cutimer.start(stream);
        call_injectcx_pass_1_kernel(
            targets, 
            input, 
            zblocks(), 
            xblocks(),
            pivots,
            active_targets, 
            num_words_major, 
            num_words_minor,
            num_qubits_padded,
            max_intermediate_blocks,
            currentblock,
            currentgrid,
            stream
        );
        if (options.sync) {
            LASTERR("failed to scan targets in pass 1");
            cutimer.stop(stream);
            LOGENDING(2, 4, "(time %.3f ms)", cutimer.time());
        } else LOGDONE(2, 4);

        // Verify pass-1 prefix.
        if (options.check_measurement) {
            checker.check_prefix_pass_1(
                targets,
                pivots,
                zblocks(), 
                xblocks(),
                active_targets,
                max_intermediate_blocks,
                pass_1_blocksize,
                pass_1_gridsize);
        }

        // Intermeditae scan of blocks resulted in pass 1.
        scan_blocks(nextPow2(pass_1_gridsize), pass_1_blocksize, stream);

        // Verify intermediate-pass prefix.
        if (options.check_measurement) {
            checker.check_prefix_intermediate_pass(
                zblocks(), 
                xblocks(),
                max_intermediate_blocks,
                pass_1_gridsize);
        }

        // Second phase of injecting CX.
        TRIM_BLOCK_IN_DEBUG_MODE(bestblockinjectfinal, bestgridinjectfinal, active_targets, num_words_minor);
        currentblock = bestblockinjectfinal, currentgrid = bestgridinjectfinal;
        if (currentblock.x > active_targets) {
            currentblock.x = active_targets == 1 ? 2 : MIN(currentblock.x, nextPow2(active_targets));
        }
        FORCE_TRIM_GRID_IN_XY(active_targets, num_words_minor);
        LOGN2(2, " Running pass-2 kernel for %d targets with block(x:%u, y:%u) and grid(x:%u, y:%u).. ", \
            active_targets, currentblock.x, currentblock.y, currentgrid.x, currentgrid.y); \
        if (options.sync) cutimer.start(stream);
        call_injectcx_pass_2_kernel(
            targets, 
            input,
            zblocks(), 
            xblocks(), 
            pivots, 
            active_targets, 
            num_words_major, 
            num_words_minor, 
            num_qubits_padded,
            max_intermediate_blocks,
            pass_1_blocksize,
            currentblock,
            currentgrid,
            stream
        );
        if (options.sync) {
            LASTERR("failed to scan targets in pass 2");
            cutimer.stop(stream);
            LOGENDING(2, 4, "(time %.3f ms)", cutimer.time());
        } else LOGDONE(2, 4);

        // Verify pass-2 prefix.
        if (options.check_measurement) {
            checker.check_prefix_pass_2(
                targets, 
                input,
                active_targets, 
                max_intermediate_blocks,
                pass_1_blocksize);
        }
    }

    void Prefix::tune_inject_cx(Tableau& input, const pivot_t* pivots, const size_t& max_active_targets) {
        assert(nextPow2(MIN_BLOCK_INTERMEDIATE_SIZE) == MIN_BLOCK_INTERMEDIATE_SIZE);
        const size_t num_qubits_padded = input.num_qubits_padded();

        // Do the first phase of prefix.
        if (options.tune_injectprepare) {
            SYNCALL;
            tune_inject_pass_1(
                bestblockinjectprepare, bestgridinjectprepare,
                2 * sizeof(word_std_t), // used to skip very large blocks.
                max_active_targets,
                num_words_minor,
                targets, 
                input, 
                zblocks(), 
                xblocks(),
                pivots,
                max_active_targets, 
                num_words_major, 
                num_words_minor,
                num_qubits_padded,
                max_intermediate_blocks
            );
            SYNCALL;
        }

        const size_t pass_1_blocksize = bestblockinjectprepare.x;
        const size_t pass_1_gridsize = ROUNDUP(max_active_targets, pass_1_blocksize);
        tune_scan_blocks(nextPow2(pass_1_gridsize), pass_1_blocksize);

        if (options.tune_injectfinal) {
            SYNCALL;
            tune_inject_pass_2(
                bestblockinjectfinal, bestgridinjectfinal,
                4 * sizeof(word_std_t),
                max_active_targets,
                num_words_minor,
                targets, 
                input,
                zblocks(), 
                xblocks(), 
                pivots, 
                max_active_targets, 
                num_words_major, 
                num_words_minor, 
                num_qubits_padded,
                max_intermediate_blocks,
                pass_1_blocksize
            );
            SYNCALL;
        }
    }

}