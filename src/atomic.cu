#include "hip/hip_runtime.h"

#include "atomic.cuh"
#include "warp.cuh"

#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

namespace QuaSARQ {

    NOINLINE_DEVICE uint32 atomicAggInc(uint32* counter) {
        coalesced_group g = coalesced_threads();
        uint32 prev;
        if (g.thread_rank() == 0) {
            prev = atomicAdd(counter, g.num_threads());
        }
        prev = g.thread_rank() + g.shfl(prev, 0);
        return prev;
    }

    NOINLINE_DEVICE uint32 atomicAggMin(uint32* min, const uint32& val) {
        const uint32 activemask = __activemask(), min_id = __ffs(activemask) - 1;
        if (laneID() == min_id)
            atomicMin(min, val);
    }

    #define EXTRACT_BYTE_FROM_ADDR(ADDR,VAL) \
	    uint64 addr_val = (uint64)ADDR; \
        uint32 al_offset = uint32(addr_val & 3) << 3; \
        uint32* byte_addr = reinterpret_cast<uint32*> (addr_val & (0xFFFFFFFFFFFFFFFCULL)); \
        uint32 byte = (VAL << al_offset) \

#if defined(WORD_SIZE_8)
    #if	defined(_DEBUG) || defined(DEBUG) || !defined(NDEBUG)
    NOINLINE_DEVICE sign_t
    #else
    NOINLINE_DEVICE void
    #endif
    atomicXOR(sign_t* addr, const uint32& value) {
        assert(value <= WORD_MAX);
		EXTRACT_BYTE_FROM_ADDR(addr, value);
        #if	defined(_DEBUG) || defined(DEBUG) || !defined(NDEBUG)
        return sign_t((atomicXor(byte_addr, byte) >> al_offset) & 0xFF);
        #else
        atomicXor(byte_addr, byte);
        #endif
    }
#else
    NOINLINE_DEVICE sign_t atomicXOR(sign_t* addr, const word_std_t& value) {
        return atomicXor(addr, value);
    }
#endif

    NOINLINE_DEVICE void atomicByteXOR(byte_t* addr, const uint32& value) {
    	EXTRACT_BYTE_FROM_ADDR(addr, value);
        atomicXor(byte_addr, byte);
    }

}