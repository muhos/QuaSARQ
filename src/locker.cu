#include "hip/hip_runtime.h"

#include "locker.cuh"

namespace QuaSARQ {

    NOINLINE_DEVICE bool DeviceLocker::unlocked() {
        assert(mutex != nullptr);
        return atomicCAS(mutex, 0, 1) == 0;
    }

    NOINLINE_DEVICE void DeviceLocker::lock() {
        assert(mutex != nullptr);
        while (atomicCAS(mutex, 0, 1) != 0);
    }

    NOINLINE_DEVICE void DeviceLocker::unlock() {
        assert(mutex != nullptr);
        atomicExch(mutex, 0);
    }

}