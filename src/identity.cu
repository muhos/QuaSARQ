#include "hip/hip_runtime.h"

#include "simulator.hpp"
#include "identity.cuh"
#include "collapse.cuh"
#include "identitycheck.cuh"

namespace QuaSARQ {

    __global__ 
    void identity_1D(IDENTITY_ARGS) {
        for_parallel_x(q, num_qubits) {
            xs->set_word_to_identity(q, column_offset);
            zs->set_word_to_identity(q, column_offset);
        }
    }

    __global__ 
    void identity_Z_1D(IDENTITY_ARGS) {
        for_parallel_x(q, num_qubits) {
            zs->set_word_to_identity(q, column_offset);
        }
    }

    __global__ 
    void identity_X_1D(IDENTITY_ARGS) {
        for_parallel_x(q, num_qubits) {
            xs->set_word_to_identity(q, column_offset);
        }
    }

    __global__ 
    void identity_extended_1D(IDENTITY_ARGS) {
        for_parallel_x(q, num_qubits) {
            xs->set_stab_to_identity(q, column_offset);
            zs->set_stab_to_identity(q, column_offset);
        }
    }

    __global__ 
    void identity_Z_extended_1D(IDENTITY_ARGS) {
        for_parallel_x(q, num_qubits) {
            xs->set_destab_to_identity(q, column_offset);
            zs->set_stab_to_identity(q, column_offset);
        }
    }

    __global__ 
    void identity_X_extended_1D(IDENTITY_ARGS) {
        for_parallel_x(q, num_qubits) {
            zs->set_destab_to_identity(q, column_offset);
            xs->set_stab_to_identity(q, column_offset);
        }
    }
    
    void Simulator::identity(
                Tableau&        tab, 
        const   size_t&         offset_per_partition, 
        const   size_t&         num_qubits_per_partition, 
        const   hipStream_t*   streams, 
        const   InitialState&   istate) 
    {
        const hipStream_t& stream = streams[KERNEL_STREAM];
        if (options.tune_identity) {
            tune_identity(
                measuring ? identity_Z_extended_1D : identity_Z_1D, 
                bestblockidentity, 
                bestgrididentity, 
                offset_per_partition, 
                num_qubits_per_partition, 
                XZ_TABLE(tab));
        }
        char state = '0';
        if (istate == Plus)
            state = '+';
        else if (istate == Imag)
            state = 'i';
        LOGN2(1, "Creating \'%c\' initial state  for size %zd and offset %zd using grid(%d) and block(%d).. ", 
            state, num_qubits_per_partition, offset_per_partition, bestgrididentity.x, bestblockidentity.x);
        if (options.sync) cutimer.start();
        if (offset_per_partition) tab.reset();
        if (measuring) { 
            if (istate == Zero)
                identity_Z_extended_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
            else if (istate == Plus)
                identity_X_extended_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
            else if (istate == Imag)
                identity_extended_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
        }
        else {
            if (istate == Zero)
                identity_Z_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
            else if (istate == Plus)
                identity_X_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
            else if (istate == Imag)
                identity_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
        }
        if (options.sync) {
            LASTERR("failed to launch identity kernel");
            CHECK(hipDeviceSynchronize());
            cutimer.stop();
            double itime = cutimer.time();
            LOG2(1, "done in %f ms.", itime);
        }
        else LOGDONE(1, 3);
        if (options.check_identity) {
            LOGN2(1, " Checking identity.. ");
            if (!check_identity(tableau, offset_per_partition, num_qubits_per_partition, measuring)) {
                LOGERROR("creating identity failed.");
            }
            LOG2(1, "%sPASSED.%s", CGREEN, CNORMAL);
        }
        if (options.print_initialtableau) 
            print_tableau(tab, MAX_DEPTH, false);
    }

}