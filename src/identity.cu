#include "hip/hip_runtime.h"

#include "simulator.hpp"
#include "identity.cuh"
#include "collapse.cuh"

namespace QuaSARQ {

    __global__ void identity_1D(const size_t column_offset, const size_t num_qubits, Table* xs, Table* zs) {
        for_parallel_x(q, num_qubits) {
            xs->set_word_to_identity(q, column_offset);
            zs->set_word_to_identity(q, column_offset);
        }
    }

    __global__ void identity_Z_1D(const size_t column_offset, const size_t num_qubits, Table* xs, Table* zs) {
        for_parallel_x(q, num_qubits) {
            zs->set_word_to_identity(q, column_offset);
        }
    }

    __global__ void identity_X_1D(const size_t column_offset, const size_t num_qubits, Table* xs, Table* zs) {
        for_parallel_x(q, num_qubits) {
            xs->set_word_to_identity(q, column_offset);
        }
    }

    __global__ void identity_extended_1D(const size_t column_offset, const size_t num_qubits, Table* xs, Table* zs) {
        for_parallel_x(q, num_qubits) {
            xs->set_stab_to_identity(q, column_offset);
            zs->set_stab_to_identity(q, column_offset);
        }
    }

    __global__ void identity_Z_extended_1D(const size_t column_offset, const size_t num_qubits, Table* xs, Table* zs) {
        for_parallel_x(q, num_qubits) {
            xs->set_destab_to_identity(q, column_offset);
            zs->set_stab_to_identity(q, column_offset);
        }
    }

    __global__ void identity_X_extended_1D(const size_t column_offset, const size_t num_qubits, Table* xs, Table* zs) {
        for_parallel_x(q, num_qubits) {
            zs->set_destab_to_identity(q, column_offset);
            xs->set_stab_to_identity(q, column_offset);
        }
    }
    
    void Simulator::identity(
                Tableau&        tab, 
        const   size_t&         offset_per_partition, 
        const   size_t&         num_qubits_per_partition, 
        const   hipStream_t*   streams, 
        const   InitialState&   istate) 
    {
        const hipStream_t& stream = streams[KERNEL_STREAM];
        if (options.tune_identity) {
            tune_identity(
                measuring ? identity_Z_extended_1D : identity_Z_1D, 
                bestblockidentity, 
                bestgrididentity, 
                offset_per_partition, 
                num_qubits_per_partition, 
                XZ_TABLE(tab));
        }
        char state = '0';
        if (istate == Plus)
            state = '+';
        else if (istate == Imag)
            state = 'i';
        LOGN2(1, "Creating \'%c\' initial state  for size %zd and offset %zd using grid(%d) and block(%d).. ", 
            state, num_qubits_per_partition, offset_per_partition, bestgrididentity.x, bestblockidentity.x);
        if (options.sync) cutimer.start();
        if (offset_per_partition) tab.reset();
        if (measuring) { 
            if (istate == Zero)
                identity_Z_extended_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
            else if (istate == Plus)
                identity_X_extended_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
            else if (istate == Imag)
                identity_extended_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
        }
        else {
            if (istate == Zero)
                identity_Z_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
            else if (istate == Plus)
                identity_X_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
            else if (istate == Imag)
                identity_1D <<< bestgrididentity, bestblockidentity, 0, stream >>> 
                    (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
        }
        if (options.sync) {
            LASTERR("failed to launch identity kernel");
            CHECK(hipDeviceSynchronize());
            cutimer.stop();
            double itime = cutimer.time();
            LOG2(1, "done in %f ms.", itime);
        }
        else LOGDONE(1, 3);
        if (options.print_initialtableau) 
            print_tableau(tab, -1, false);
    }

}