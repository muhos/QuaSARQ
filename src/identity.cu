#include "hip/hip_runtime.h"

#include "simulator.hpp"
#include "identity.cuh"
#include "collapse.cuh"
#include "tuner.cuh"

namespace QuaSARQ {

    // Set these to the tuned values (other than 1) 
    // to avoid trigeering the tuner. 
    dim3 bestBlockReset(96), bestGridReset(693);
    dim3 bestBlockIdentity(96), bestGridIdentity(693);

#ifdef INTERLEAVE_XZ

    __global__ void identity_1D(const size_t column_offset, const size_t num_qubits, Table* ps) {
        for_parallel_x(q, num_qubits) {
            ps->set_x_word_to_identity(q, column_offset);
            ps->set_z_word_to_identity(q, column_offset);
        }
    }

    __global__ void identity_Z_1D(const size_t column_offset, const size_t num_qubits, Table* ps) {
        for_parallel_x(q, num_qubits) {
            ps->set_z_word_to_identity(q, column_offset);
        }
    }

    __global__ void identity_X_1D(const size_t column_offset, const size_t num_qubits, Table* ps) {
        for_parallel_x(q, num_qubits) {
            ps->set_x_word_to_identity(q, column_offset);
        }
    }

#else

    __global__ void identity_1D(const size_t column_offset, const size_t num_qubits, Table* xs, Table* zs) {
        for_parallel_x(q, num_qubits) {
            xs->set_word_to_identity(q, column_offset);
            zs->set_word_to_identity(q, column_offset);
        }
    }

    __global__ void identity_Z_1D(const size_t column_offset, const size_t num_qubits, Table* zs) {
        for_parallel_x(q, num_qubits) {
            zs->set_word_to_identity(q, column_offset);
        }
    }

    __global__ void identity_X_1D(const size_t column_offset, const size_t num_qubits, Table* xs) {
        for_parallel_x(q, num_qubits) {
            xs->set_word_to_identity(q, column_offset);
        }
    }

    __global__ void identity_extended_1D(const size_t column_offset, const size_t num_qubits, Table* xs, Table* zs) {
        for_parallel_x(q, num_qubits) {
            xs->set_word_to_identity(q, column_offset, num_qubits);
            zs->set_word_to_identity(q, column_offset, num_qubits);
        }
    }

    __global__ void identity_Z_extended_1D(const size_t column_offset, const size_t num_qubits, Table* xs, Table* zs) {
        for_parallel_x(q, num_qubits) {
            xs->set_word_to_identity(q, column_offset);
            zs->set_word_to_identity(q, column_offset, num_qubits);
        }
    }

    __global__ void identity_X_extended_1D(const size_t column_offset, const size_t num_qubits, Table* xs, Table* zs) {
        for_parallel_x(q, num_qubits) {
            zs->set_word_to_identity(q, column_offset);
            xs->set_word_to_identity(q, column_offset, num_qubits);
        }
    }

#endif
    
    void Simulator::identity(Tableau<DeviceAllocator>& tab, const size_t& offset_per_partition, const size_t& num_qubits_per_partition, const hipStream_t* streams, const InitialState& istate) {
        assert(num_qubits_per_partition <= tab.num_qubits_padded());
        if (options.tune_identity) {
            tune_kernel(identity_Z_1D, "Identity", bestBlockIdentity, bestGridIdentity, offset_per_partition, num_qubits_per_partition, Z_TABLE(tab));
        }
        char state = '0';
        if (istate == Plus)
            state = '+';
        else if (istate == Imag)
            state = 'i';
        LOGN2(1, "Creating \'%c\' initial state  for size %zd and offset %zd using grid(%d) and block(%d).. ", state, num_qubits_per_partition, offset_per_partition, bestGridIdentity.x, bestBlockIdentity.x);
        if (options.sync) cutimer.start();
        if (offset_per_partition) tab.reset();
        if (measuring) { 
            if (istate == Zero)
                identity_Z_extended_1D <<< bestGridIdentity, bestBlockIdentity, 0, streams[KERNEL_STREAM] >>> (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
            else if (istate == Plus)
                identity_X_extended_1D <<< bestGridIdentity, bestBlockIdentity, 0, streams[KERNEL_STREAM] >>> (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
            else if (istate == Imag)
                identity_extended_1D <<< bestGridIdentity, bestBlockIdentity, 0, streams[KERNEL_STREAM] >>> (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
        }
        else {
            if (istate == Zero)
                identity_Z_1D <<< bestGridIdentity, bestBlockIdentity, 0, streams[KERNEL_STREAM] >>> (offset_per_partition, num_qubits_per_partition, Z_TABLE(tab));
            else if (istate == Plus)
                identity_X_1D <<< bestGridIdentity, bestBlockIdentity, 0, streams[KERNEL_STREAM] >>> (offset_per_partition, num_qubits_per_partition, X_TABLE(tab));
            else if (istate == Imag)
                identity_1D <<< bestGridIdentity, bestBlockIdentity, 0, streams[KERNEL_STREAM] >>> (offset_per_partition, num_qubits_per_partition, XZ_TABLE(tab));
        }
        if (options.sync) {
            LASTERR("failed to launch identity kernel");
            cutimer.stop();
            double itime = cutimer.time();
            LOG2(1, "done in %f ms.", itime);
        }
        else LOGDONE(1, 3);
        if (options.print_initial_tableau) 
            print_tableau(tab, -1, false);
    }

}